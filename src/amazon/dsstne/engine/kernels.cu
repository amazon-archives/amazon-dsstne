#include "hip/hip_runtime.h"
/*


   Copyright 2016  Amazon.com, Inc. or its affiliates. All Rights Reserved.

   Licensed under the Apache License, Version 2.0 (the "License"). You may not use this file except in compliance with the License. A copy of the License is located at

   http://aws.amazon.com/apache2.0/

   or in the "license" file accompanying this file. This file is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the License for the specific language governing permissions and limitations under the License.
 */

#include "GpuTypes.h"
#include "NNTypes.h"
#include <limits>

static __constant__ GpuData cData;

void SetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cData), &(getGpu()._data), sizeof(GpuData));     
    RTERROR(status, "hipMemcpyToSymbol: SetKernelsGpuData copy to cData failed");
}

void GetKernelsGpuData()
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&(getGpu()._data), cData, sizeof(GpuData));     
    RTERROR(status, "hipMemcpyFromSymbol: GetKernelsGpuData copy From cData failed");
}


uint32_t CalculateBlocks(uint64_t size)
{
    return (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
}

// Scales and biases a weight matrix previously generated
__global__ void
LAUNCH_BOUNDS()
kScaleAndBias_kernel(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint64_t offset             = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset < size)
    {
        NNFloat value           = pData[offset];
        pData[offset]           = scale * value - bias;
    }
}

void kScaleAndBias(NNFloat* pData, uint64_t size, NNFloat scale, NNFloat bias)
{
    uint32_t blocks             = CalculateBlocks(size);
    kScaleAndBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pData, size, scale, bias);
    LAUNCHERROR("kScaleAndBias_kernel");
}


// Initializes hidden or output unit with bias of single incoming unit
__global__ void
LAUNCH_BOUNDS()
kClearUnit_kernel(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint64_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]              = pBias[bpos];
    }
}


void kClearUnit(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t batch)
{
    uint64_t size               = (uint64_t)stride * (uint64_t)batch;
    uint32_t blocks             = CalculateBlocks(size);
    kClearUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias, stride, size);
    LAUNCHERROR("kClearUnit_kernel");
}

// Initializes hidden or output unit with biases of 2 incoming units
__global__ void
LAUNCH_BOUNDS()
kClearDualSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]              = pBias1[bpos] + pBias2[bpos];
    }
}

void kClearDualSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size               = (uint64_t)stride * (uint64_t)batch;
    uint32_t blocks             = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kClearDualSourceUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);
    LAUNCHERROR("kClearDualSourceUnit_kernel");
}



// Initializes hidden or output unit with biases of 3 incoming units
__global__ void
LAUNCH_BOUNDS()
kClearTripleSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]              = pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

void kClearTripleSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size               = (uint64_t)stride * (uint64_t)batch;
    uint32_t blocks             = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kClearTripleSourceUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kClearTripleSource_kernel");
}

// Initializes hidden or output unit with biases of 4 incoming units
__global__ void
LAUNCH_BOUNDS()
kClearQuadSourceUnit_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]              = pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

void kClearQuadSourceUnit(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size               = (uint64_t)stride * (uint64_t)batch;
    uint32_t blocks             = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kClearQuadSourceUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kClearQuadSource_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kLoadSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {
        uint32_t pos1                   = pos + position;                            
        pos1                            = cData._bShuffleIndices ?  cData._pShuffleIndex[pos1] : pos1;
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            uint64_t pos2               = offset + pSparseIndex[start];
            pUnit[pos2]                 = w;
            start                      += cData._warpSize;
        }
    }
}

void kLoadSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadSparseInputUnit failed");
    kLoadSparseInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadSparseInputUnit_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kLoadIndexedSparseInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {
        uint32_t pos1                   = pos + position;                            
        pos1                            = pIndex[cData._bShuffleIndices ?  cData._pShuffleIndex[pos1] : pos1];
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            uint64_t pos2               = offset + pSparseIndex[start];
            pUnit[pos2]                 = w;
            start                      += cData._warpSize;
        }
    }
}

void kLoadIndexedSparseInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadIndexedSparseInputUnit failed");
    kLoadIndexedSparseInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight);
    LAUNCHERROR("kLoadpIndexedSparseInputUnit_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kLoadSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {
        uint32_t pos1                   = pos + position;                            
        pos1                            = cData._bShuffleIndices ?  cData._pShuffleIndex[pos1] : pos1;
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            uint64_t pos2               = offset + pSparseIndex[start];
            T data                      = pSparseData[start];
            pUnit[pos2]                 = w * data;
            start                      += cData._warpSize;
        }
    }
}

template<typename T>
void kLoadSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadSparseAnalogInputUnit failed");    
    kLoadSparseAnalogInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadSparseAnalogInputUnit_kernel");
}


template<typename T>
__global__ void
LAUNCH_BOUNDS()
kLoadIndexedSparseAnalogInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {
        uint32_t pos1                   = pos + position;                            
        pos1                            = pIndex[cData._bShuffleIndices ?  cData._pShuffleIndex[pos1] : pos1];
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = (pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0;
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            uint64_t pos2               = offset + pSparseIndex[start];
            T data                      = pSparseData[start];
            pUnit[pos2]                 = w * data;
            start                      += cData._warpSize;
        }
    }
}

template<typename T>
void kLoadIndexedSparseAnalogInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadIndexedSparseAnalogInputUnit failed");    
    kLoadIndexedSparseAnalogInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData);
    LAUNCHERROR("kLoadIndexedSparseAnalogInputUnit_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kLoadSparseDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {                           
        uint32_t pos1                   = cData._bShuffleIndices ?  cData._pShuffleIndex[pos + position] : pos + position;
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0);
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            NNFloat value               = pRandom[start];
            uint64_t pos2               = offset + pSparseIndex[start];
            if (value >= cData._denoising_p)
                pUnit[pos2]             = w;
            start                      += cData._warpSize;
        }
    }
}


void kLoadSparseDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    
    
    //printf("KLSPDU %u %u %u %u %lu %lu %lu %lu %lu\n", position, batch, stride, blocks, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pRandom);

    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadSparseDenoisedInputUnit failed");
    kLoadSparseDenoisedInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom);
    LAUNCHERROR("kLoadSparseDenoisedInputUnit_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kLoadIndexedSparseDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {                           
        uint32_t pos1                   = pIndex[cData._bShuffleIndices ?  cData._pShuffleIndex[pos + position] : pos + position];
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0);
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            NNFloat value               = pRandom[start];
            uint64_t pos2               = offset + pSparseIndex[start];
            if (value >= cData._denoising_p)
                pUnit[pos2]             = w;
            start                      += cData._warpSize;
        }
    }
}


void kLoadIndexedSparseDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadIndexedSparseDenoisedInputUnit failed");
    kLoadIndexedSparseDenoisedInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom);
    LAUNCHERROR("kLoadIndexedSparseDenoisedInputUnit_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kLoadSparseAnalogDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {                           
        uint32_t pos1                   = cData._bShuffleIndices ?  cData._pShuffleIndex[pos + position] : pos + position;
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0);
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            NNFloat value               = pRandom[start];
            uint64_t pos2               = offset + pSparseIndex[start];
            T data                      = pSparseData[start];
            if (value >= cData._denoising_p)
                pUnit[pos2]             = w * data;
            start                      += cData._warpSize;
        }
    }
}

template<typename T>
void kLoadSparseAnalogDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T *pSparseData, NNFloat* pRandom)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadSparseAnalogDenoisedInputUnit failed");
    kLoadSparseAnalogDenoisedInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom);
    LAUNCHERROR("kLoadSparseAnalogDenoisedInputUnit_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kLoadIndexedSparseAnalogDenoisedInputUnit_kernel(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom)
{
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) / cData._warpSize;
    if (pos < batch)
    {                           
        uint32_t pos1                   = pIndex[cData._bShuffleIndices ?  cData._pShuffleIndex[pos + position] : pos + position];
        uint64_t start                  = pSparseStart[pos1] + (threadIdx.x & cData._warpMask);
        uint64_t end                    = pSparseEnd[pos1];
        NNFloat w                       = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[pos1] : (NNFloat)1.0);
        uint64_t offset                 = pos * stride;
        while (start < end)
        {
            NNFloat value               = pRandom[start];
            uint64_t pos2               = offset + pSparseIndex[start];
            T data                      = pSparseData[start];
            if (value >= cData._denoising_p)
                pUnit[pos2]             = w * data;
            start                      += cData._warpSize;
        }
    }
}

template<typename T>
void kLoadIndexedSparseAnalogDenoisedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t *pSparseIndex, NNFloat* pDataWeight, T *pSparseData, NNFloat* pRandom)
{
    uint32_t last                       = position + batch;
    uint32_t count                      = last - position;
    uint32_t blocks                     = (count * getGpu()._warpSize + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;

    hipError_t status                  = hipMemset(pUnit, 0, (uint64_t)batch * (uint64_t)stride * sizeof(NNFloat));
    RTERROR(status, "kLoadIndexedSparseAnalogDenoisedInputUnit failed");
    kLoadIndexedSparseAnalogDenoisedInputUnit_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, stride, pUnit, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom);
    LAUNCHERROR("kLoadIndexedSparseAnalogDenoisedInputUnit_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kLoadInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, T* pData)
{
    uint64_t pos                        = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1                   = cData._bShuffleIndices ?  cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position;
        uint64_t soffset                = pos1 * stride + pos;
        uint64_t doffset                = blockIdx.x * stride + pos;
        pUnit[doffset]                  = pData[soffset];
    }
}

__global__ void
kLoadNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, unsigned char* pData)
{
    uint64_t pos                        = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1                   = cData._bShuffleIndices ?  cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position;
        uint64_t soffset                = pos1 * stride + pos;
        uint64_t doffset                = blockIdx.x * stride + pos;
        pUnit[doffset]                  = (NNFloat)pData[soffset] * (NNFloat)(1.0 / 256.0) - (NNFloat)0.5;
    }
}

__global__ void
kLoadNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, char* pData)
{
    uint64_t pos          = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1                   = cData._bShuffleIndices ?  cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position;
        uint64_t soffset                = pos1 * stride + pos;
        uint64_t doffset                = blockIdx.x * stride + pos;
        pUnit[doffset]                  = (NNFloat)pData[soffset] * (NNFloat)(1.0 / 128.0);
    }
}

template<typename T> void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, T* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    LAUNCHERROR("kLoadInputUnit_kernel");
}

template<> void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, unsigned char* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    LAUNCHERROR("kLoadNormalizedInputUnit_kernel");
}

template<> void kLoadInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, char* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pData);
    LAUNCHERROR("kLoadNormalizedInputUnit_kernel");
}


template<typename T>
__global__ void
LAUNCH_BOUNDS()
kLoadIndexedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData)
{
    uint64_t pos                        = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1                   = pIndex[cData._bShuffleIndices ?  cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position];
        uint64_t soffset                = pos1 * stride + pos;
        uint64_t doffset                = blockIdx.x * stride + pos;
        pUnit[doffset]                  = pData[soffset];
    }
}

__global__ void
kLoadIndexedNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData)
{
    uint64_t pos                        = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1                   = pIndex[cData._bShuffleIndices ?  cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position];
        uint64_t soffset                = pos1 * stride + pos;
        uint64_t doffset                = blockIdx.x * stride + pos;
        pUnit[doffset]                  = (NNFloat)pData[soffset] * (NNFloat)(1.0 / 256.0) - (NNFloat)0.5;
    }
}

__global__ void
kLoadIndexedNormalizedInputUnit_kernel(uint32_t position, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData)
{
    uint64_t pos          = (blockIdx.y * blockDim.x) + threadIdx.x;
    if (pos < stride)
    {
        uint32_t pos1                   = pIndex[cData._bShuffleIndices ?  cData._pShuffleIndex[blockIdx.x + position] : blockIdx.x + position];
        uint64_t soffset                = pos1 * stride + pos;
        uint64_t doffset                = blockIdx.x * stride + pos;
        pUnit[doffset]                  = (NNFloat)pData[soffset] * (NNFloat)(1.0 / 128.0);
    }
}

template<typename T> void kLoadIndexedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, T* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadIndexedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData);
    LAUNCHERROR("kLoadIndexedInputUnit_kernel");
}

template<> void kLoadIndexedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, unsigned char* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadIndexedNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData);
    LAUNCHERROR("kLoadIndexedNormalizedInputUnit_kernel");
}

template<> void kLoadIndexedInputUnit(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pUnit, uint32_t* pIndex, char* pData)
{
    dim3 grid(batch, (stride + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);
    kLoadIndexedNormalizedInputUnit_kernel<<<grid, getGpu()._threadsPerBlock>>>(position, stride, pUnit, pIndex, pData);
    LAUNCHERROR("kLoadIndexedNormalizedInputUnit_kernel");
}

// Adds bias from single incoming unit
__global__ void
LAUNCH_BOUNDS()
kAddBias_kernel(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t size)
{
    uint32_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]             += pBias[bpos];
    }
}


void kAddBias(NNFloat* pUnit, NNFloat* pBias, uint32_t stride, uint32_t batch)
{
    uint32_t size               = stride * batch;
    uint32_t blocks             = CalculateBlocks(size);
    kAddBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias, stride, size);
    LAUNCHERROR("kAddBias_kernel");
}


// Adds biases of 2 incoming units to hidden or output unit
__global__ void
LAUNCH_BOUNDS()
kAddDualBias_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]             += pBias1[bpos] + pBias2[bpos];
    }
}

void kAddDualBias(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, uint32_t stride, uint32_t batch)
{
    uint64_t size               = (uint64_t)stride * (uint64_t)batch;
    uint32_t blocks             = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kAddDualBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, stride, size);
    LAUNCHERROR("kAddDualBias_kernel");
}

// Adds biases of 3 incoming units to hidden or output unit
__global__ void
LAUNCH_BOUNDS()
kAddTripleBias_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]             += pBias1[bpos] + pBias2[bpos] + pBias3[pos];
    }
}

void kAddTripleBias(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, uint32_t stride, uint32_t batch)
{
    uint64_t size               = (uint64_t)stride * (uint64_t)batch;
    uint32_t blocks             = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kAddTripleBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, stride, size);
    LAUNCHERROR("kAddTripleBias_kernel");
}

// Adds biases of 4 incoming units to hidden or output unit
__global__ void
LAUNCH_BOUNDS()
kAddQuadBias_kernel(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t size)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bpos               = pos % stride;
    if (pos < size)
    {
        pUnit[pos]             += pBias1[bpos] + pBias2[bpos] + pBias3[pos] + pBias4[pos];
    }
}

void kAddQuadBias(NNFloat* pUnit, NNFloat* pBias1, NNFloat* pBias2, NNFloat* pBias3, NNFloat* pBias4, uint32_t stride, uint32_t batch)
{
    uint64_t size               = (uint64_t)stride * (uint64_t)batch;
    uint32_t blocks             = (size + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock;
    kAddQuadBias_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pBias1, pBias2, pBias3, pBias4, stride, size);
    LAUNCHERROR("kAddQuadBias_kernel");
}

#if (__CUDA_ARCH__ >= 600)
static const uint32_t MAXSPARSE = SM_6X_MAXSPARSE;
static const uint32_t MAXSPARSEANALOG = SM_6X_MAXSPARSEANALOG;
#elif (__CUDA_ARCH__ >= 500)
static const uint32_t MAXSPARSE = SM_5X_MAXSPARSE;
static const uint32_t MAXSPARSEANALOG = SM_5X_MAXSPARSEANALOG;
#else
static const uint32_t MAXSPARSE = SM_3X_MAXSPARSE;
static const uint32_t MAXSPARSEANALOG = SM_3X_MAXSPARSEANALOG;
#endif


__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSE];                         // Shared set of offsets to non-zero weights

    // Read sparse indices into shared memory so they're only read once
    position                        = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start                  = pSparseStart[position];
    uint64_t end                    = pSparseEnd[position];
    NNFloat w                       = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                          += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSE);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]            = pSparseIndex[tstart] * stride;
            pos                    += blockDim.x;
            tstart                 += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += w * pWeight[offset + opos];
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}


void kCalculateSparseZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pUnit, beta);
    LAUNCHERROR("kCalculateSparseZ_kernel");
}

__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSE];                         // Shared set of offsets to non-zero weights

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSE);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]            = pSparseIndex[tstart] * stride;
            pos                    += blockDim.x;
            tstart                 += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += w * pWeight[offset + opos];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}


void kCalculateIndexedSparseZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseZ_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseAnalogZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ T sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseIndex[tstart] * stride;
            sValue[pos]         = w * pSparseData[tstart];
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseAnalogZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseIndex[tstart] * stride;
            sValue[pos]         = w * ((NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 256.0));
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseAnalogZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseIndex[tstart] * stride;
            sValue[pos]         = w * ((NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 256.0));
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<typename T> void kCalculateSparseAnalogZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseAnalogZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pUnit, beta);
    LAUNCHERROR("kCalculateSparseAnalogZ_kernel");
}

/*
template<> void kCalculateSparseAnalogZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseAnalogZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pUnit, beta);
    LAUNCHERROR("kCalculateSparseAnalogZ_kernel");
}

template<> void kCalculateSparseAnalogZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseAnalogZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pUnit, beta);
    LAUNCHERROR("kCalculateSparseAnalogZ_kernel");
}
*/

template<typename T>
__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseAnalogZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ T sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseIndex[tstart] * stride;
            sValue[pos]         = w * pSparseData[tstart];
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseAnalogZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseIndex[tstart] * stride;
            sValue[pos]         = w * ((NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 256.0));
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseAnalogZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseIndex[tstart] * stride;
            sValue[pos]         = w * ((NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 128.0));
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    unit           += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<typename T> void kCalculateIndexedSparseAnalogZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseAnalogZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseAnalogZ_kernel");
}

/*
template<> void kCalculateIndexedSparseAnalogZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseAnalogZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseAnalogZ_kernel");
}

template<> void kCalculateIndexedSparseAnalogZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseAnalogZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseAnalogZ_kernel");
}
*/

__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSE];                         // Shared set of offsets to non-zero weights

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            NNFloat value           = pRandom[tstart];
            sOffset[pos]            = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            pos                    += blockDim.x;
            tstart                 += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos];
                }

                // Write output
                pUnit[opos]         = w * unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

void kCalculateSparseDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateSparseDenoisedZ_kernel");
}

__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSE];                         // Shared set of offsets to non-zero weights

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;

    while (start < end)
    {
        sOpos                       = blockDim.x;
        uint32_t inputs             = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend               = start + inputs;
        uint64_t tstart             = start + threadIdx.x;
        uint32_t pos                = threadIdx.x;

        while (tstart < tend)
        {
            NNFloat value           = pRandom[tstart];
            sOffset[pos]            = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            pos                    += blockDim.x;
            tstart                 += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos];
                }

                // Write output
                pUnit[opos]         = w * unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

void kCalculateIndexedSparseDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseDenoisedZ_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseAnalogDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ T sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;

        while (tstart < tend)
        {
            NNFloat value       = pRandom[tstart];
            sOffset[pos]        = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            sValue[pos]         = pSparseData[tstart] * w;
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : pUnit[opos];
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseAnalogDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ int32_t sOffset[MAXSPARSEANALOG];                    // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;
        while (tstart < tend)
        {
            NNFloat value       = pRandom[tstart];
            sOffset[pos]        = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            sValue[pos]         = (NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 256.0) * w;
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseAnalogDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x;    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;
        while (tstart < tend)
        {
            NNFloat value       = pRandom[tstart];
            sOffset[pos]        = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            sValue[pos]         = (NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 128.0) * w;
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<typename T> void kCalculateSparseAnalogDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseAnalogDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateSparseAnalogDenoisedZ_kernel");
}

/*
template<> void kCalculateSparseAnalogDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseAnalogDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateSparseAnalogDenoisedZ_kernel");
}

template<> void kCalculateSparseAnalogDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseAnalogDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateSparseAnalogDenoisedZ_kernel");
}
*/

template<typename T>
__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseAnalogDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ T sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;

        while (tstart < tend)
        {
            NNFloat value       = pRandom[tstart];
            sOffset[pos]        = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            sValue[pos]         = pSparseData[tstart] * w;
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseAnalogDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ int32_t sOffset[MAXSPARSEANALOG];                    // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;

        while (tstart < tend)
        {
            NNFloat value       = pRandom[tstart];
            sOffset[pos]        = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            sValue[pos]         = (NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 256.0) * w;
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<>
__global__ void
LAUNCH_BOUNDS256()
kCalculateIndexedSparseAnalogDenoisedZ_kernel(uint32_t position, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read sparse indices into shared memory so they're only read once
    sOpos                       = blockDim.x;
    position                    = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + blockIdx.x] : position + blockIdx.x];    
    uint64_t start              = pSparseStart[position];
    uint64_t end                = pSparseEnd[position];
    NNFloat w                   = cData._denoising_q * ((pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0);
    pUnit                      += blockIdx.x * stride;
    while (start < end)
    {
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;

        while (tstart < tend)
        {
            NNFloat value       = pRandom[tstart];
            sOffset[pos]        = (value < cData._denoising_p) ? cData._maxUint32_t : (int32_t)pSparseIndex[tstart] * stride;
            sValue[pos]         = (NNFloat)pSparseData[tstart] * (NNFloat)(1.0 / 128.0) * w;
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t tgx                = threadIdx.x & cData._warpMask;    
        uint32_t opos               = threadIdx.x - tgx;
        while (opos < stride)
        {        
            // Read all non-zero inputs
            opos                   += tgx;
            if (opos < stride)
            {
                NNFloat unit        = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : (beta * pUnit[opos]);
                for (uint32_t i = 0; i < inputs; i++)
                {
                    uint32_t offset = sOffset[i];
                    if (offset != cData._maxUint32_t)
                        unit       += pWeight[offset + opos] * sValue[i];  
                }

                // Write output
                pUnit[opos]         = unit;
            }
            opos                   -= tgx;

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos                = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                    = SHFL(opos, 0);
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
        }
        beta                    = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
    }
}

template<typename T> void kCalculateIndexedSparseAnalogDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseAnalogDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseAnalogDenoisedZ_kernel");
}

/*
template<> void kCalculateIndexedSparseAnalogDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseAnalogDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseAnalogDenoisedZ_kernel");
}

template<> void kCalculateIndexedSparseAnalogDenoisedZ(uint32_t position, uint32_t batch, uint32_t stride, NNFloat* pWeight, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat* pRandom, NNFloat* pUnit, NNFloat beta)
{
    uint32_t threads            = min(256, ((stride + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateIndexedSparseAnalogDenoisedZ_kernel<<<batch, threads>>>(position, stride, pWeight, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pUnit, beta);
    LAUNCHERROR("kCalculateIndexedSparseAnalogDenoisedZ_kernel");
}
*/

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseTransposedMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        while (start < end)
        {
            uint32_t index                  = pSparseIndex[start];
            uint32_t opos                   = atomicAdd(&pSparseTransposedEnd[index], 1);
            pSparseTransposedIndex[opos]    = bpos;
            start                          += cData._warpSize;                   
        }
    }
}

__global__ void
LAUNCH_BOUNDS()
kCalculateWeightedSparseTransposedMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = pDataWeight[position];
        while (start < end)
        {
            uint32_t index                  = pSparseIndex[start];
            uint32_t opos                   = atomicAdd(&pSparseTransposedEnd[index], 1);
            pSparseTransposedIndex[opos]    = bpos;
            pSparseTransposedData[opos]     = w;
            start                          += cData._warpSize;                   
        }
    }
}

void kCalculateSparseTransposedMatrix(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    if (pDataWeight == NULL)
    {
        kCalculateSparseTransposedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pSparseStart, pSparseEnd, pSparseIndex, pSparseTransposedEnd, pSparseTransposedIndex);
        LAUNCHERROR("kCalculateSparseTransposedMatrix_kernel");
    }
    else
    {
        kCalculateWeightedSparseTransposedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
        LAUNCHERROR("kCalculateWeightedSparseTransposedMatrix_kernel");
    }
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseTransposedMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        while (start < end)
        {
            uint32_t index                  = pSparseIndex[start];
            uint32_t opos                   = atomicAdd(&pSparseTransposedEnd[index], 1);
            pSparseTransposedIndex[opos]    = bpos;
            start                          += cData._warpSize;                   
        }
    }
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedWeightedSparseTransposedMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = pDataWeight[position];
        while (start < end)
        {
            uint32_t index                  = pSparseIndex[start];
            uint32_t opos                   = atomicAdd(&pSparseTransposedEnd[index], 1);
            pSparseTransposedIndex[opos]    = bpos;
            pSparseTransposedData[opos]     = w;
            start                          += cData._warpSize;                   
        }
    }
}

void kCalculateIndexedSparseTransposedMatrix(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch);
    if (pDataWeight == NULL)
    {
        kCalculateIndexedSparseTransposedMatrix_kernel<<<batch, getGpu()._warpSize>>>(position, batch, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pSparseTransposedEnd, pSparseTransposedIndex);
        LAUNCHERROR("kCalculateIndexedSparseTransposedMatrix_kernel");
    }
    else
    {
        kCalculateIndexedWeightedSparseTransposedMatrix_kernel<<<batch, getGpu()._warpSize>>>(position, batch, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
        LAUNCHERROR("kCalculateIndexedWeightedSparseTransposedMatrix_kernel");
    }
}

__global__ void
LAUNCH_BOUNDS()
kCalculateSparseTransposedDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
            }
            start                          += cData._warpSize;                   
        }
    }
}

__global__ void
LAUNCH_BOUNDS()
kCalculateWeightedSparseTransposedDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = cData._denoising_q * pDataWeight[position];
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w;
            }
            start                          += cData._warpSize;                   
        }
    }
}

void kCalculateSparseTransposedDenoisedMatrix(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    if (pDataWeight == NULL)
    {
        kCalculateSparseTransposedDenoisedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pSparseStart, pSparseEnd, pSparseIndex, pRandom, pSparseTransposedEnd, pSparseTransposedIndex);
        LAUNCHERROR("kCalculateSparseTransposedDenoisedMatrix_kernel");
    }
    else
    {
        kCalculateWeightedSparseTransposedDenoisedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
        LAUNCHERROR("kCalculateWeightedSparseTransposedDenoisedMatrix_kernel");
    }
}


__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseTransposedDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
            }
            start                          += cData._warpSize;                   
        }
    }
}

__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedWeightedSparseTransposedDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = cData._denoising_q * pDataWeight[position];
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w;
            }
            start                          += cData._warpSize;                   
        }
    }
}

void kCalculateIndexedSparseTransposedDenoisedMatrix(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, NNFloat* pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    if (pDataWeight == NULL)
    {
        kCalculateIndexedSparseTransposedDenoisedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pRandom, pSparseTransposedEnd, pSparseTransposedIndex);
        LAUNCHERROR("kCalculateIndexedSparseTransposedDenoisedMatrix_kernel");
    }
    else
    {
        kCalculateIndexedWeightedSparseTransposedDenoisedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pRandom, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
        LAUNCHERROR("kCalculateIndexedWeightedSparseTransposedDenoisedMatrix_kernel")
    }
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseTransposedAnalogMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            uint32_t index                  = pSparseIndex[start];
            T value                         = pSparseData[start];
            uint32_t opos                   = atomicAdd(&pSparseTransposedEnd[index], 1);
            pSparseTransposedIndex[opos]    = bpos;
            pSparseTransposedData[opos]     = w * value;
            start                          += cData._warpSize;                   
        }
    }
}

template<typename T>
void kCalculateSparseTransposedAnalogMatrix(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateSparseTransposedAnalogMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
    LAUNCHERROR("kCalculateSparseTransposedAnalogMatrix_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseTransposedAnalogMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];    
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            uint32_t index                  = pSparseIndex[start];
            T value                         = pSparseData[start];
            uint32_t opos                   = atomicAdd(&pSparseTransposedEnd[index], 1);
            pSparseTransposedIndex[opos]    = bpos;
            pSparseTransposedData[opos]     = w * value;
            start                          += cData._warpSize;                   
        }
    }
}

template<typename T>
void kCalculateIndexedSparseTransposedAnalogMatrix(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateIndexedSparseTransposedAnalogMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
    LAUNCHERROR("kCalculateIndexedSparseTransposedAnalogMatrix_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseTransposedAnalogDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                T value                     = pSparseData[start];
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w * value;
            }
            start                          += cData._warpSize;                   
        }
    }
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseTransposedAnalogDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                NNFloat value               = (NNFloat)pSparseData[start] * (NNFloat)(1.0 / 256.0);
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w * value;
            }
            start                          += cData._warpSize;                   
        }
    }
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateSparseTransposedAnalogDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos;
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                NNFloat value               = (NNFloat)pSparseData[start] * (NNFloat)(1.0 / 128.0);
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w * value;
            }
            start                          += cData._warpSize;                   
        }
    }
}

template<typename T>
void kCalculateSparseTransposedAnalogDenoisedMatrix(uint32_t position, uint32_t batch, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateSparseTransposedAnalogDenoisedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
    LAUNCHERROR("kCalculateSparseTransposedAnalogDenoisedMatrix_kernel");
}

template<typename T>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseTransposedAnalogDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                T value                     = pSparseData[start];
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w * value;
            }
            start                          += cData._warpSize;                   
        }
    }
}
template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseTransposedAnalogDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, unsigned char* pSparseData, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                NNFloat value               = (NNFloat)pSparseData[start] * (NNFloat)(1.0 / 256.0);
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w * value;
            }
            start                          += cData._warpSize;                   
        }
    }
}

template<>
__global__ void
LAUNCH_BOUNDS()
kCalculateIndexedSparseTransposedAnalogDenoisedMatrix_kernel(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, char* pSparseData, NNFloat *pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    // Determine batch position
    uint32_t bpos                           = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                            = threadIdx.x & cData._warpMask;
    
    // Add indices to sparse transposed activation matrix
    if (bpos < batch)
    {
        position                            = pIndex[cData._bShuffleIndices ? cData._pShuffleIndex[position + bpos] : position + bpos];
        uint64_t start                      = pSparseStart[position] + tgx;
        uint64_t end                        = pSparseEnd[position];
        NNFloat w                           = (pDataWeight != NULL) ? pDataWeight[position] : (NNFloat)1.0;
        while (start < end)
        {
            NNFloat rnd                     = pRandom[start];
            uint32_t index                  = pSparseIndex[start];
            if (rnd >= cData._denoising_p)
            {
                NNFloat value               = (NNFloat)pSparseData[start] * (NNFloat)(1.0 / 128.0);
                uint32_t opos               = atomicAdd(&pSparseTransposedEnd[index], 1);
                pSparseTransposedIndex[opos]= bpos;
                pSparseTransposedData[opos] = w * value;
            }
            start                          += cData._warpSize;                   
        }
    }
}

template<typename T>
void kCalculateIndexedSparseTransposedAnalogDenoisedMatrix(uint32_t position, uint32_t batch, uint32_t* pIndex, uint64_t* pSparseStart, uint64_t* pSparseEnd, uint32_t* pSparseIndex, NNFloat* pDataWeight, T* pSparseData, NNFloat* pRandom, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData)
{
    uint32_t blocks             = CalculateBlocks(batch * getGpu()._warpSize);
    kCalculateIndexedSparseTransposedAnalogDenoisedMatrix_kernel<<<blocks, getGpu()._threadsPerBlock>>>(position, batch, pIndex, pSparseStart, pSparseEnd, pSparseIndex, pDataWeight, pSparseData, pRandom, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData);
    LAUNCHERROR("kCalculateIndexedSparseTransposedAnalogDenoisedMatrix_kernel");
}


__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseTransposedWeightGradient_kernel(NNFloat alpha, NNFloat beta, uint32_t n, uint32_t* pSparseTransposedStart, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pDelta, NNFloat* pWeightGradient)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSE];                         // Shared set of offsets to non-zero weights

    // Read transposed sparse indices into shared memory so they're only read once
    uint64_t start              = pSparseTransposedStart[blockIdx.x];
    uint64_t end                = pSparseTransposedEnd[blockIdx.x];
    alpha                      *= cData._denoising_q;
    pWeightGradient            += blockIdx.x * n;
    do
    {
        sOpos                   = blockDim.x;         
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSE);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseTransposedIndex[tstart] * n;
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions
        uint32_t opos           = threadIdx.x;
        uint32_t tgx            = threadIdx.x & cData._warpMask;    
        while (opos < n)
        {        
            // Read all non-zero inputs, accumulate in 64-bit FP to maintain deterministic results
            NNFloat oldgradient = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : beta * pWeightGradient[opos];
            int64_t sum         = 0;
            for (uint32_t i = 0; i < inputs; i++)
            {
                uint32_t offset = sOffset[i];
                sum            += llrintf(ERRORSCALEF * pDelta[offset + opos]);  
            }

            // Write output
            NNFloat fsum        = alpha * (NNFloat)((double)sum * ONEOVERERRORSCALE);
            pWeightGradient[opos] = oldgradient + fsum;            

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos            = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                = SHFL(opos, 0);
            opos               += tgx;
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
            beta                = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
        }
    }
    while (start < end);
}


void kCalculateSparseTransposedWeightGradient(NNFloat alpha, NNFloat beta, uint32_t m, uint32_t n, uint32_t* pSparseTransposedStart, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pDelta, NNFloat* pWeightGradient)
{
    uint32_t threads            = min(256, ((m + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);
    kCalculateSparseTransposedWeightGradient_kernel<<<m, threads>>>(alpha, beta, n, pSparseTransposedStart, pSparseTransposedEnd, pSparseTransposedIndex, pDelta, pWeightGradient);
    LAUNCHERROR("kCalculateSparseTransposedWeightGradient_kernel");
}

__global__ void
LAUNCH_BOUNDS256()
kCalculateSparseTransposedAnalogWeightGradient_kernel(NNFloat alpha, NNFloat beta, uint32_t n, uint32_t* pSparseTransposedStart, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData, NNFloat* pDelta, NNFloat* pWeightGradient)
{
__shared__ uint32_t sOpos;                                      // Shared output position
__shared__ uint32_t sOffset[MAXSPARSEANALOG];                   // Shared set of offsets to non-zero weights
__shared__ NNFloat sValue[MAXSPARSEANALOG];

    // Read transposed sparse indices and data into shared memory so they're only read once 
    uint64_t start              = pSparseTransposedStart[blockIdx.x];
    uint64_t end                = pSparseTransposedEnd[blockIdx.x];
    alpha                      *= cData._denoising_q;
    pWeightGradient            += blockIdx.x * n;
    do
    {
        sOpos                   = blockDim.x;        
        uint32_t inputs         = ullmin(end - start, (uint64_t)MAXSPARSEANALOG);
        uint64_t tend           = start + inputs;
        uint64_t tstart         = start + threadIdx.x;
        uint32_t pos            = threadIdx.x;

        while (tstart < tend)
        {
            sOffset[pos]        = pSparseTransposedIndex[tstart] * n;
            sValue[pos]         = pSparseTransposedData[start];
            pos                += blockDim.x;
            tstart             += blockDim.x;
        }

        __threadfence();
        __syncthreads();

        // Cycle through all output positions

        uint32_t opos           = threadIdx.x;
        uint32_t tgx            = threadIdx.x & cData._warpMask;    
        while (opos < n)
        {        
            // Read all non-zero inputs, accumulate in 64-bit FP to maintain deterministic results
            NNFloat oldgradient = (beta == (NNFloat)0.0) ? (NNFloat)0.0 : beta * pWeightGradient[opos];
            int64_t sum         = 0;
            for (uint32_t i = 0; i < inputs; i++)
            {
                uint32_t offset = sOffset[i];
                NNFloat value   = sValue[i]; 
                sum            += llrintf(ERRORSCALEF * value * pDelta[offset + opos]);  
            }

            // Write output
            NNFloat fsum        = alpha * (NNFloat)((double)sum * ONEOVERERRORSCALE);
            pWeightGradient[opos] = oldgradient + fsum;            

            // Advance to next set of outputs
            if (tgx == 0)
            {
                opos            = atomicAdd(&sOpos, cData._warpSize);
            }
            opos                = SHFL(opos, 0);
            opos               += tgx;
        }

        // Advance to next block of sparse inputs, syncing if necessary
        start                   = tend;
        if (start < end)
        {
            __threadfence();
            __syncthreads();
            beta                = (NNFloat)1.0;             // Set beta to 1.0 for any remaining gradient accumulation
        }
    }
    while (start < end);
}

void kCalculateSparseTransposedAnalogWeightGradient(NNFloat alpha, NNFloat beta, uint32_t m, uint32_t n, uint32_t* pSparseTransposedStart, uint32_t* pSparseTransposedEnd, uint32_t* pSparseTransposedIndex, NNFloat* pSparseTransposedData, NNFloat* pDelta, NNFloat* pWeightGradient)
{
    uint32_t threads            = min(256, ((m + getGpu()._warpSize - 1) >> getGpu()._warpBits) << getGpu()._warpBits);    
    kCalculateSparseTransposedAnalogWeightGradient_kernel<<<m, threads>>>(alpha, beta, n, pSparseTransposedStart, pSparseTransposedEnd, pSparseTransposedIndex, pSparseTransposedData, pDelta, pWeightGradient);
    LAUNCHERROR("kCalculateSparseTransposedAnalogWeightGradient_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kUpdateBiases_kernel(NNFloat alpha, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBias)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum             = (NNFloat)0.0;
        pDelta                 += pos;
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                += *pDelta;
            pDelta             += width;
        }
        pBias[pos]             -= alpha * sum;
    }
}

void kUpdateBiases(NNFloat alpha, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, batch, width, pDelta, pBias);
    LAUNCHERROR("kUpdateBiases_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateRegularizationError_kernel(NNFloat* pWeight, uint64_t size, NNFloat lambda, NNFloat lambda1)
{
    uint64_t pos                = (blockIdx.x * blockDim.x) + threadIdx.x;
    NNFloat error               = (NNFloat)0.0;
    if (pos < size)
    {
        NNFloat w               = pWeight[pos];
        error                   = lambda * w * w + lambda1 * abs(w);   
    }

    // Reduce error across threads
    REDUCEERROR(error)
}

// Calculates raw weight decay/regularization error
NNFloat kCalculateRegularizationError(NNFloat lambda, NNFloat lambda1, NNFloat* pWeight, uint64_t size)
{
    uint32_t blocks         = CalculateBlocks(size);
    hipMemset(getGpu()._data._pAccumulator, 0, sizeof(uint64_t));
    kCalculateRegularizationError_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pWeight, size, (NNFloat)0.5 * lambda, lambda1);
    LAUNCHERROR("kCalculateRegularizationError_kernel");
    getGpu()._pbAccumulator->Download(); 
    return (NNFloat)((double)(getGpu()._pbAccumulator->_pSysData[0]) * ONEOVERERRORSCALE);    
}

__global__ void
LAUNCH_BOUNDS()
kSGDUpdateWeights_kernel(NNFloat alpha, NNFloat lambda, NNFloat lambda1, uint64_t size, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat g               = pWeightGradient[pos];
        NNFloat w               = pWeight[pos];
        pWeight[pos]            = w + alpha * (g - lambda * w - lambda1 * sgn(w));
    }
}

void kSGDUpdateWeights(NNFloat alpha, NNFloat lambda, NNFloat lambda1, uint64_t size, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint32_t blocks             = CalculateBlocks(size);
    kSGDUpdateWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, lambda, lambda1, size, pWeightGradient, pWeight);
    LAUNCHERROR("kSGDUpdateWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kSGDUpdateBiases_kernel(NNFloat alpha, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBias)
{
    uint32_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum             = 0.0f;
        pDelta                 += pos;

        // Calculate bias gradient
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                += *pDelta;
            pDelta             += width;
        }
        sum                    /= (NNFloat)batch;

        // Update velocity and bias
        NNFloat bias            = pBias[pos];
        pBias[pos]              = bias - alpha * sum;
    }
}

void kSGDUpdateBiases(NNFloat alpha, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kSGDUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, batch, width, pDelta, pBias);
    LAUNCHERROR("kSGDUpdateBiases_kernel");
}


__global__ void
LAUNCH_BOUNDS()
kMomentumUpdateWeights_kernel(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat g               = pWeightGradient[pos];
        NNFloat w               = pWeight[pos];
        NNFloat v               = pWeightVelocity[pos];
        v                       = mu * v + alpha * (g - lambda * w - lambda1 * sgn(w));
        pWeightVelocity[pos]    = v;
        pWeight[pos]            = w + v;
    }
}

void kMomentumUpdateWeights(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint32_t blocks             = CalculateBlocks(size);
    kMomentumUpdateWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, lambda, lambda1, mu, size, pWeightVelocity, pWeightGradient, pWeight);
    LAUNCHERROR("kMomentumUpdateWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kMomentumUpdateBiases_kernel(NNFloat alpha, NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint32_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum             = 0.0f;
        pDelta                 += pos;

        // Calculate bias gradient
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                += *pDelta;
            pDelta             += width;
        }
        sum                    /= (NNFloat)batch;

        // Update velocity and bias
        NNFloat v               = pBiasVelocity[pos];
        v                       = mu * v - alpha * sum;
        pBiasVelocity[pos]      = v;
        pBias[pos]             += v;
    }
}

void kMomentumUpdateBiases(NNFloat alpha, NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kMomentumUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, mu, batch, width, pDelta, pBiasVelocity, pBias);
    LAUNCHERROR("kMomentumUpdateBiases_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAdaGradUpdateWeights_kernel(NNFloat alpha, NNFloat lambda, NNFloat lambda1, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat g               = pWeightGradient[pos];
        NNFloat w               = pWeight[pos];
        NNFloat v               = pWeightVelocity[pos];
        g                      -= lambda * w + lambda1 * sgn(w);
        v                      += g * g;
        pWeightVelocity[pos]    = v;
        pWeight[pos]            = w + alpha * g * rsqrt(max(0.000000001f, v));
    }
}

void kAdaGradUpdateWeights(NNFloat alpha, NNFloat lambda, NNFloat lambda1, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    unsigned long blocks        = CalculateBlocks(size);
    kAdaGradUpdateWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, lambda, lambda1, size, pWeightVelocity, pWeightGradient, pWeight);
    LAUNCHERROR("kAdaGradUpdateWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAdaGradUpdateBiases_kernel(NNFloat alpha, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum             = 0.0f;
        pDelta                 += pos;

        // Calculate bias gradient
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                += *pDelta;
            pDelta             += width;
        }
        sum                    /= (NNFloat)batch;

        // Update velocity and bias
        NNFloat v               = pBiasVelocity[pos];
        v                      += sum * sum;
        pBiasVelocity[pos]      = v;
        pBias[pos]             -= alpha * sum * rsqrt(max(0.000000001f, v));
    }
}

void kAdaGradUpdateBiases(NNFloat alpha, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kAdaGradUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, batch, width, pDelta, pBiasVelocity, pBias);
    LAUNCHERROR("kAdaGradUpdateBiases_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAdaDeltaUpdateWeights_kernel(NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeightGradientVelocity, NNFloat* pWeight)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat g                       = pWeightGradient[pos];
        NNFloat w                       = pWeight[pos];
        NNFloat v                       = pWeightVelocity[pos];
        NNFloat vg                      = pWeightGradientVelocity[pos];
        g                              -= lambda * w + lambda1 * sgn(w);
        vg                              = mu * vg + ((NNFloat)1.0 - mu) * g * g;
        NNFloat dw                      = sqrt(max((NNFloat)0.000000001, v) / max((NNFloat)0.000000001, vg)) * g;
        v                               = mu * v + ((NNFloat)1.0 - mu) * dw * dw;
        pWeightVelocity[pos]            = v;
        pWeightGradientVelocity[pos]    = vg;
        pWeight[pos]                    = w + dw;
    }
}

void kAdaDeltaUpdateWeights(NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeightGradientVelocity, NNFloat* pWeight)
{
    unsigned long blocks        = CalculateBlocks(size);
    kAdaDeltaUpdateWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(lambda, lambda1, mu, size, pWeightVelocity, pWeightGradient, pWeightGradientVelocity, pWeight);
    LAUNCHERROR("kAdaDeltaUpdateWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAdaDeltaUpdateBiases_kernel(NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBiasGradientVelocity, NNFloat* pBias)
{
    uint64_t pos                    = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum                 = (NNFloat)0.0;
        pDelta                     += pos;

        // Calculate bias gradient
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                    += *pDelta;
            pDelta                 += width;
        }
        sum                        /= (NNFloat)batch;

        // Update velocity and bias
        NNFloat v                   = pBiasVelocity[pos];
        NNFloat vg                  = pBiasGradientVelocity[pos];        
        vg                          = mu * vg + ((NNFloat)1.0 - mu) * sum * sum;
        NNFloat dw                  = sqrt(max((NNFloat)0.000000001, v) / max((NNFloat)0.000000001, vg)) * sum;        
        v                           = mu * v + ((NNFloat)1.0 - mu) * dw * dw;
        pBiasVelocity[pos]          = v;
        pBiasGradientVelocity[pos]  = vg;        
        pBias[pos]                 -= dw;
    }
}

void kAdaDeltaUpdateBiases(NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBiasGradientVelocity, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kAdaDeltaUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(mu, batch, width, pDelta, pBiasVelocity, pBiasGradientVelocity, pBias);
    LAUNCHERROR("kAdaDeltaUpdateBiases_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAdamUpdateWeights_kernel(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat beta1, NNFloat beta2, NNFloat t, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeightGradientVelocity, NNFloat* pWeight)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat dw                      = pWeightGradient[pos];
        NNFloat w                       = pWeight[pos];
        NNFloat vdw                     = pWeightVelocity[pos];
        NNFloat sdw                     = pWeightGradientVelocity[pos];
        dw                             -= lambda * w + lambda1 * sgn(w);
        vdw                             = beta1 * vdw + ((NNFloat)1.0 - beta1) * dw;
        sdw                             = beta2 * sdw + ((NNFloat)1.0 - beta2) * dw * dw;
        t                              += (NNFloat)1.0;
        pWeightVelocity[pos]            = vdw;
        pWeightGradientVelocity[pos]    = sdw;
        vdw                            /= (NNFloat)1.0 - pow(beta1, t);
        sdw                            /= (NNFloat)1.0 - pow(beta2, t);        
        dw                              = alpha * vdw / (sqrt(sdw) + (NNFloat)1.0e-8);
        pWeight[pos]                    = w + dw;
    }
}

void kAdamUpdateWeights(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat beta1, NNFloat beta2, NNFloat t, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeightGradientVelocity, NNFloat* pWeight)
{
    unsigned long blocks        = CalculateBlocks(size);
    kAdamUpdateWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, lambda, lambda1, beta1, beta2, t, size, pWeightVelocity, pWeightGradient, pWeightGradientVelocity, pWeight);
    LAUNCHERROR("kAdamUpdateWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAdamUpdateBiases_kernel(NNFloat alpha, NNFloat beta1, NNFloat beta2, NNFloat t, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBiasGradientVelocity, NNFloat* pBias)
{
    uint64_t pos                    = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum                 = (NNFloat)0.0;
        pDelta                     += pos;

        // Calculate bias gradient
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                    += *pDelta;
            pDelta                 += width;
        }
        sum                        /= (NNFloat)batch;

        // Update velocity and bias
        NNFloat vdw                 = pBiasVelocity[pos];
        NNFloat sdw                 = pBiasGradientVelocity[pos];
        vdw                         = beta1 * vdw + ((NNFloat)1.0 - beta1) * sum;
        sdw                         = beta2 * sdw + ((NNFloat)1.0 - beta2) * sum * sum;
        t                          += (NNFloat)1.0;
        pBiasVelocity[pos]          = vdw;
        pBiasGradientVelocity[pos]  = sdw;
        vdw                        /= (NNFloat)1.0 - pow(beta1, t);
        sdw                        /= (NNFloat)1.0 - pow(beta2, t);        
        NNFloat dw                  = alpha * vdw / (sqrt(sdw) + (NNFloat)1.0e-8);
        pBias[pos]                 -= dw;
    }
}

void kAdamUpdateBiases(NNFloat alpha, NNFloat mu, NNFloat mu1, NNFloat t, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBiasGradientVelocity, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kAdamUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, mu, mu1, t, batch, width, pDelta, pBiasVelocity, pBiasGradientVelocity, pBias);
    LAUNCHERROR("kAdamUpdateBiases_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kNesterovUpdateWeights_kernel(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat g               = pWeightGradient[pos];
        NNFloat w               = pWeight[pos];
        NNFloat vOld            = pWeightVelocity[pos];
        NNFloat vNew            = mu * vOld + alpha * (g - lambda * w - lambda1 * sgn(w));
        pWeightVelocity[pos]    = vNew;
        w                       = w + vNew + mu * (vNew - vOld);
        pWeight[pos]            = w;      
    }
}

void kNesterovUpdateWeights(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint32_t blocks             = CalculateBlocks(size);
    kNesterovUpdateWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, lambda, lambda1, mu, size, pWeightVelocity, pWeightGradient, pWeight);
    LAUNCHERROR("kNesterovUpdateWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kNesterovUpdateBiases_kernel(NNFloat alpha, NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum             = 0.0f;
        pDelta                 += pos;

        // Calculate bias gradient
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                += *pDelta;
            pDelta             += width;
        }
        sum                    /= (NNFloat)batch;

        // Update velocity and bias
        NNFloat vOld            = pBiasVelocity[pos];
        NNFloat vNew            = mu * vOld - alpha * sum;
        pBiasVelocity[pos]      = vNew;
        pBias[pos]             += vNew + mu * (vNew - vOld);
    }
}

void kNesterovUpdateBiases(NNFloat alpha, NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kNesterovUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, mu, batch, width, pDelta, pBiasVelocity, pBias);
    LAUNCHERROR("kNesterovUpdateBiases_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kNesterovShiftWeights_kernel(NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeight)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat w               = pWeight[pos];
        NNFloat v               = pWeightVelocity[pos];
        pWeight[pos]            = w + mu * v;
    }
}

void kNesterovShiftWeights(NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeight)
{
    uint32_t blocks             = CalculateBlocks(size);
    kNesterovShiftWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(mu, size, pWeightVelocity, pWeight);
    LAUNCHERROR("kNesterovShiftWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kNesterovShiftBiases_kernel(NNFloat mu, uint32_t width, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat b               = pBias[pos];
        NNFloat v               = pBiasVelocity[pos];
        pBias[pos]              = b + mu * v;
    }
}

void kNesterovShiftBiases(NNFloat mu, uint32_t width, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kNesterovShiftBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(mu, width, pBiasVelocity, pBias);
    LAUNCHERROR("kNesterovShiftBiases_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kRMSPropUpdateWeights_kernel(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint64_t pos  = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat g               = pWeightGradient[pos];
        NNFloat w               = pWeight[pos];
        NNFloat v               = pWeightVelocity[pos];
        g                      -= lambda * w + lambda1 * sgn(w);
        v                       = mu * v + (1.0f - mu) * g * g;
        pWeightVelocity[pos]    = v;
        pWeight[pos]            = w + alpha * g * rsqrt(max(0.000000001f, v));
    }
}

void kRMSPropUpdateWeights(NNFloat alpha, NNFloat lambda, NNFloat lambda1, NNFloat mu, uint64_t size, NNFloat* pWeightVelocity, NNFloat* pWeightGradient, NNFloat* pWeight)
{
    uint32_t blocks             = CalculateBlocks(size);
    kRMSPropUpdateWeights_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, lambda, lambda1, mu, size, pWeightVelocity, pWeightGradient, pWeight);
    LAUNCHERROR("kRMSPropUpdateWeights_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kRMSPropUpdateBiases_kernel(NNFloat alpha, NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint64_t pos                = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < width)
    {
        NNFloat sum             = 0.0f;
        pDelta                 += pos;

        // Calculate bias gradient
        for (uint32_t i = 0; i < batch; i++)
        {
            sum                += *pDelta;
            pDelta             += width;
        }
        sum                    /= (NNFloat)batch;

        // Update velocity and bias
        NNFloat v               = pBiasVelocity[pos];
        v                       = mu * v + (1.0f - mu) * sum * sum;
        pBiasVelocity[pos]      = v;
        pBias[pos]             -= alpha * sum * rsqrt(max(0.000000001f, v));
    }
}

void kRMSPropUpdateBiases(NNFloat alpha, NNFloat mu, uint32_t batch, uint32_t width, NNFloat* pDelta, NNFloat* pBiasVelocity, NNFloat* pBias)
{
    uint32_t blocks             = CalculateBlocks(width);
    kRMSPropUpdateBiases_kernel<<<blocks, getGpu()._threadsPerBlock>>>(alpha, mu, batch, width, pDelta, pBiasVelocity, pBias);
    LAUNCHERROR("kRMSPropUpdateBiases_kernel");
}

#include "bitonic.h"
__global__ void
LAUNCH_BOUNDS()
kCalculateTopK_32_kernel(NNFloat* pOutputBuffer, NNFloat* pKeyBuffer, uint32_t* pValueBuffer, uint32_t batch, uint32_t width, uint32_t k)
{
__shared__ volatile NNFloat sKey[64 * 4];
__shared__ volatile uint32_t sValue[64 * 4];


    uint32_t pos                    = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                    = threadIdx.x & cData._warpMask;
            
    if (pos < batch)
    {
        NNFloat *pOutput            = pOutputBuffer + pos * width;
        uint32_t offset             = threadIdx.x >> cData._warpBits;
        volatile NNFloat* psKey     = &sKey[64 * offset];
        volatile uint32_t* psValue  = &sValue[64 * offset];

        // Initialize values to 
        NNFloat k0                  = -MAX_VALUE;
        NNFloat k1                  = -MAX_VALUE;
        uint32_t v0                 = 0;
        uint32_t v1                 = 0;

        // Read first 32 elements into registers
        uint32_t wpos               = tgx;
        if (wpos < width)
        {
            k0                      = pOutput[wpos];
            v0                      = wpos;
        }
        wpos                       += cData._warpSize;

        // Run through remainder of data
        NNFloat minValue            = -MAX_VALUE;
        uint32_t rpos               = 32;
        uint32_t bufferSize         = 0;
        NNFloat key1, key2;
        uint32_t value1, value2;
        uint32_t otgx;
        bool flag;
        while (rpos < width)
        {
            // Read block of data
            unsigned wpos           = rpos + tgx;
            NNFloat key             = -MAX_VALUE;
            uint32_t value          = wpos;
            if (wpos < width)
            {
                key                 = pOutput[wpos];                
            }
            
            // Add values > minValue to shared memory buffer
            uint32_t count          = BALLOT(key > minValue);
            if (key > minValue)
            {
                uint32_t mask       = 0xffffffff >> (32 - tgx);
                uint32_t offset     = __popc(count & mask);
                offset             += bufferSize;
                psKey[offset]       = key;
                psValue[offset]     = value;
            }
            bufferSize             += __popc(count);

            // Check if buffer is full
            if (bufferSize >= 32)
            {
                // Sort 64 elements
                k1                  = psKey[tgx];
                v1                  = psValue[tgx];
                bool flag;
                BITONICSORT64_64();

                // Set minValue to the new min in the warp register queue
                // Registers are sorted in descending order k0 > k1 > k2 ... > k_n
                minValue = k1;

                // Shift members in shared memory to beginning
                bufferSize         -= 32;
                if (tgx < bufferSize)
                {
                    psKey[tgx]      = psKey[tgx + 32];
                    psValue[tgx]    = psValue[tgx + 32];
                }
            }

            // Advance to next block of data
            rpos                    += cData._warpSize;
        }

        // Do final sort if buffer has any remaining data
        if ((bufferSize > 0) || (width <= 32))
        {
            // Store sentinel values in registers
            k1                       = -MAX_VALUE;
            v1                       = 0;

            // Load last block of unsorted data into registers
            if (tgx < bufferSize)
            {
                k1                   = psKey[tgx];
                v1                   = psValue[tgx];
            }
            BITONICSORT64_64();
        }

        // Copy results to key and value pointers
        NNFloat* pKey                = pKeyBuffer + pos * k;
        uint32_t* pValue             = pValueBuffer + pos * k;                
        wpos                         = tgx;
        if (wpos < k)
        {
            pKey[wpos]               = k0;
            pValue[wpos]             = v0;
        }
        wpos                        += cData._warpSize;
    }
}


__global__ void
LAUNCH_BOUNDS()
kCalculateTopK_64_kernel(NNFloat* pOutputBuffer, NNFloat* pKeyBuffer, uint32_t* pValueBuffer, uint32_t batch, uint32_t width, uint32_t k)
{
__shared__ volatile NNFloat sKey[96 * 4];
__shared__ volatile uint32_t sValue[96 * 4];


    uint32_t pos                    = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                    = threadIdx.x & cData._warpMask;
            
    if (pos < batch)
    {
        NNFloat *pOutput            = pOutputBuffer + pos * width;
        uint32_t offset             = threadIdx.x >> cData._warpBits;
        volatile NNFloat* psKey     = &sKey[96 * offset];
        volatile uint32_t* psValue  = &sValue[96 * offset];

        // Initialize values to 
        NNFloat k0                  = -MAX_VALUE;
        NNFloat k1                  = -MAX_VALUE;
        NNFloat k2                  = -MAX_VALUE;
        NNFloat k3                  = -MAX_VALUE;
        uint32_t v0                 = 0;
        uint32_t v1                 = 0;
        uint32_t v2                 = 0;
        uint32_t v3                 = 0;

        // Read first 64 elements into registers
        uint32_t wpos               = tgx;
        if (wpos < width)
        {
            k0                      = pOutput[wpos];
            v0                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k1                      = pOutput[wpos];
            v1                      = wpos;
        }
        wpos                       += cData._warpSize;

     
        // Run through remainder of data
        NNFloat minValue            = -MAX_VALUE;
        uint32_t rpos               = 64;
        uint32_t bufferSize         = 0;
        NNFloat key1, key2;
        uint32_t value1, value2;
        uint32_t otgx;
        bool flag;
        while (rpos < width)
        {
            // Read block of data
            unsigned wpos           = rpos + tgx;
            NNFloat key             = -MAX_VALUE;
            uint32_t value          = wpos;
            if (wpos < width)
            {
                key                 = pOutput[wpos];                
            }
            
            // Add values > minValue to shared memory buffer
            uint32_t count          = BALLOT(key > minValue);
            if (key > minValue)
            {
                uint32_t mask       = 0xffffffff >> (32 - tgx);
                uint32_t offset     = __popc(count & mask);
                offset             += bufferSize;
                psKey[offset]       = key;
                psValue[offset]     = value;
            }
            bufferSize             += __popc(count);

            // Check if buffer is full
            if (bufferSize >= 64)
            {
                // Sort 128 elements
                k2                  = psKey[tgx];
                v2                  = psValue[tgx];
                k3                  = psKey[tgx + cData._warpSize];
                v3                  = psValue[tgx + cData._warpSize];
                bool flag;
                BITONICSORT128_128();

                // Set minValue to the new min in the warp register queue
                // Registers are sorted in descending order k0 > k1 > k2 ... > k_n
                minValue = k3;

                // Shift members in shared memory to beginning
                bufferSize         -= 64;
                if (tgx < bufferSize)
                {
                    psKey[tgx]      = psKey[tgx + 64];
                    psValue[tgx]    = psValue[tgx + 64];
                }
            }

            // Advance to next block of data
            rpos                    += cData._warpSize;
        }

        // Do final sort if buffer has any remaining data
        if ((bufferSize > 0) || (width <= 64))
        {
            // Store sentinel values in registers
            k2                       = -MAX_VALUE;
            k3                       = -MAX_VALUE;
            v2                       = 0;
            v3                       = 0;

            // Load last block of unsorted data into registers
            if (tgx < bufferSize)
            {
                k2                   = psKey[tgx];
                v2                   = psValue[tgx];
            }
            if (tgx + cData._warpSize < bufferSize)
            {
                k3                   = psKey[tgx + cData._warpSize];
                v3                   = psValue[tgx + cData._warpSize];
            }

            BITONICSORT128_128();
        }

        // Copy results to key and value pointers
        NNFloat* pKey                = pKeyBuffer + pos * k;
        uint32_t* pValue             = pValueBuffer + pos * k;                
        wpos                         = tgx;
        if (wpos < k)
        {
            pKey[wpos]               = k0;
            pValue[wpos]             = v0;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k1;
            pValue[wpos]             = v1;
        }
        wpos                        += cData._warpSize;
    }
}

__global__ void
LAUNCH_BOUNDS()
kCalculateTopK_128_kernel(NNFloat* pOutputBuffer, NNFloat* pKeyBuffer, uint32_t* pValueBuffer, uint32_t batch, uint32_t width, uint32_t k)
{
__shared__ volatile NNFloat sKey[160 * 4];
__shared__ volatile uint32_t sValue[160 * 4];


    uint32_t pos                    = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                    = threadIdx.x & cData._warpMask;
        
    
    if (pos < batch)
    {
        NNFloat *pOutput            = pOutputBuffer + pos * width;
        uint32_t offset             = threadIdx.x >> cData._warpBits;
        volatile NNFloat* psKey     = &sKey[160 * offset];
        volatile uint32_t* psValue  = &sValue[160 * offset];

        // Initialize values to 
        NNFloat k0                  = -MAX_VALUE;
        NNFloat k1                  = -MAX_VALUE;
        NNFloat k2                  = -MAX_VALUE;
        NNFloat k3                  = -MAX_VALUE;
        NNFloat k4                  = -MAX_VALUE;
        NNFloat k5                  = -MAX_VALUE;
        NNFloat k6                  = -MAX_VALUE;
        NNFloat k7                  = -MAX_VALUE;
        uint32_t v0                 = 0;
        uint32_t v1                 = 0;
        uint32_t v2                 = 0;
        uint32_t v3                 = 0;
        uint32_t v4                 = 0;
        uint32_t v5                 = 0;
        uint32_t v6                 = 0;
        uint32_t v7                 = 0;

        // Read first 128 elements into registers
        uint32_t wpos               = tgx;
        if (wpos < width)
        {
            k0                      = pOutput[wpos];
            v0                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k1                      = pOutput[wpos];
            v1                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k2                      = pOutput[wpos];
            v2                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k3                      = pOutput[wpos];
            v3                      = wpos;
        }
     
        // Run through remainder of data
        NNFloat minValue            = -MAX_VALUE;
        uint32_t rpos               = 128;
        uint32_t bufferSize         = 0;
        NNFloat key1, key2;
        uint32_t value1, value2;
        uint32_t otgx;
        bool flag;
        while (rpos < width)
        {
            // Read block of data
            unsigned wpos           = rpos + tgx;
            NNFloat key             = -MAX_VALUE;
            uint32_t value          = wpos;
            if (wpos < width)
            {
                key                 = pOutput[wpos];                
            }
            
            // Add values > minValue to shared memory buffer
            uint32_t count          = BALLOT(key > minValue);
            if (key > minValue)
            {
                uint32_t mask       = 0xffffffff >> (32 - tgx);
                uint32_t offset     = __popc(count & mask);
                offset             += bufferSize;
                psKey[offset]       = key;
                psValue[offset]     = value;
            }
            bufferSize             += __popc(count);

            // Check if buffer is full
            if (bufferSize >= 128)
            {
                // Sort 256 elements
                k4                  = psKey[tgx];
                v4                  = psValue[tgx];
                k5                  = psKey[tgx + cData._warpSize];
                v5                  = psValue[tgx + cData._warpSize];
                k6                  = psKey[tgx + 2 * cData._warpSize];
                v6                  = psValue[tgx + 2 * cData._warpSize];
                k7                  = psKey[tgx + 3 * cData._warpSize];
                v7                  = psValue[tgx + 3 * cData._warpSize];
                bool flag;
                BITONICSORT256_256();

                // Set minValue to the new min in the warp register queue
                // Registers are sorted in descending order k0 > k1 > k2 ... > k_n
                minValue = k7;

                // Shift members in shared memory to beginning
                bufferSize         -= 128;
                if (tgx < bufferSize)
                {
                    psKey[tgx]      = psKey[tgx + 128];
                    psValue[tgx]    = psValue[tgx + 128];
                }
            }

            // Advance to next block of data
            rpos                    += cData._warpSize;
        }

        // Do final sort if buffer has any remaining data
        if ((bufferSize > 0) || (width <= 128))
        {
            // Store sentinel values in registers
            k4                       = -MAX_VALUE;
            k5                       = -MAX_VALUE;
            k6                       = -MAX_VALUE;
            k7                       = -MAX_VALUE;
            v4                       = 0;
            v5                       = 0;
            v6                       = 0;
            v7                       = 0;

            // Load last block of unsorted data into registers
            if (tgx < bufferSize)
            {
                k4                   = psKey[tgx];
                v4                   = psValue[tgx];
            }
            if (tgx + cData._warpSize < bufferSize)
            {
                k5                   = psKey[tgx + cData._warpSize];
                v5                   = psValue[tgx + cData._warpSize];
            }
            if (tgx + 2 * cData._warpSize < bufferSize)
            {
                k6                   = psKey[tgx + 2 * cData._warpSize];
                v6                   = psValue[tgx + 2 * cData._warpSize];
            }
            if (tgx + 3 * cData._warpSize < bufferSize)
            {
                k7                   = psKey[tgx + 3 * cData._warpSize];
                v7                   = psValue[tgx + 3 * cData._warpSize];
            }

            BITONICSORT256_256();
        }

        // Copy results to key and value pointers
        NNFloat* pKey                = pKeyBuffer + pos * k;
        uint32_t* pValue             = pValueBuffer + pos * k;                
        wpos                         = tgx;
        if (wpos < k)
        {
            pKey[wpos]               = k0;
            pValue[wpos]             = v0;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k1;
            pValue[wpos]             = v1;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k2;
            pValue[wpos]             = v2;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k3;
            pValue[wpos]             = v3;
        }
    }
}


__global__ void
LAUNCH_BOUNDS()
kCalculateTopK_256_kernel(NNFloat* pOutputBuffer, NNFloat* pKeyBuffer, uint32_t* pValueBuffer, uint32_t batch, uint32_t width, uint32_t k)
{
__shared__ volatile NNFloat sKey[288 * 4];
__shared__ volatile uint32_t sValue[288 * 4];


    uint32_t pos                    = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                    = threadIdx.x & cData._warpMask;
        
    
    if (pos < batch)
    {
        NNFloat *pOutput            = pOutputBuffer + pos * width;
        uint32_t offset             = threadIdx.x >> cData._warpBits;
        volatile NNFloat* psKey     = &sKey[288 * offset];
        volatile uint32_t* psValue  = &sValue[288 * offset];

        // Initialize values to 
        NNFloat k0                  = -MAX_VALUE;
        NNFloat k1                  = -MAX_VALUE;
        NNFloat k2                  = -MAX_VALUE;
        NNFloat k3                  = -MAX_VALUE;
        NNFloat k4                  = -MAX_VALUE;
        NNFloat k5                  = -MAX_VALUE;
        NNFloat k6                  = -MAX_VALUE;
        NNFloat k7                  = -MAX_VALUE;
        NNFloat k8                  = -MAX_VALUE;
        NNFloat k9                  = -MAX_VALUE;
        NNFloat k10                 = -MAX_VALUE;
        NNFloat k11                 = -MAX_VALUE;
        NNFloat k12                 = -MAX_VALUE;
        NNFloat k13                 = -MAX_VALUE;
        NNFloat k14                 = -MAX_VALUE;
        NNFloat k15                 = -MAX_VALUE;
        uint32_t v0                 = 0;
        uint32_t v1                 = 0;
        uint32_t v2                 = 0;
        uint32_t v3                 = 0;
        uint32_t v4                 = 0;
        uint32_t v5                 = 0;
        uint32_t v6                 = 0;
        uint32_t v7                 = 0;
        uint32_t v8                 = 0;
        uint32_t v9                 = 0;
        uint32_t v10                = 0;
        uint32_t v11                = 0;
        uint32_t v12                = 0;
        uint32_t v13                = 0;
        uint32_t v14                = 0;
        uint32_t v15                = 0;
        
        // Read first 256 elements into registers
        uint32_t wpos               = tgx;
        if (wpos < width)
        {
            k0                      = pOutput[wpos];
            v0                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k1                      = pOutput[wpos];
            v1                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k2                      = pOutput[wpos];
            v2                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k3                      = pOutput[wpos];
            v3                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k4                      = pOutput[wpos];
            v4                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k5                      = pOutput[wpos];
            v5                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k6                      = pOutput[wpos];
            v6                      = wpos;
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k7                      = pOutput[wpos];
            v7                      = wpos;
        }
             
        // Run through remainder of data
        NNFloat minValue            = -MAX_VALUE;
        uint32_t rpos               = 256;
        uint32_t bufferSize         = 0;
        NNFloat key1, key2;
        uint32_t value1, value2;
        uint32_t otgx;
        bool flag;
        while (rpos < width)
        {
            // Read block of data
            unsigned wpos           = rpos + tgx;
            NNFloat key             = -MAX_VALUE;
            uint32_t value          = wpos;
            if (wpos < width)
            {
                key                 = pOutput[wpos];                
            }
            
            // Add values > minValue to shared memory buffer
            uint32_t count          = BALLOT(key > minValue);
            if (key > minValue)
            {
                uint32_t mask       = 0xffffffff >> (32 - tgx);
                uint32_t offset     = __popc(count & mask);
                offset             += bufferSize;
                psKey[offset]       = key;
                psValue[offset]     = value;
            }
            bufferSize             += __popc(count);

            // Check if buffer is full
            if (bufferSize >= 256)
            {
                // Sort 512 elements
                k8                  = psKey[tgx];
                v8                  = psValue[tgx];
                k9                  = psKey[tgx + cData._warpSize];
                v9                  = psValue[tgx + cData._warpSize];
                k10                 = psKey[tgx + 2 * cData._warpSize];
                v10                 = psValue[tgx + 2 * cData._warpSize];
                k11                 = psKey[tgx + 3 * cData._warpSize];
                v11                 = psValue[tgx + 3 * cData._warpSize];
                k12                 = psKey[tgx + 4 * cData._warpSize];
                v12                 = psValue[tgx + 4 * cData._warpSize];
                k13                 = psKey[tgx + 5 * cData._warpSize];
                v13                 = psValue[tgx + 5 * cData._warpSize];
                k14                 = psKey[tgx + 6 * cData._warpSize];
                v14                 = psValue[tgx + 6 * cData._warpSize];                
                k15                 = psKey[tgx + 7 * cData._warpSize];
                v15                 = psValue[tgx + 7 * cData._warpSize];
                bool flag;
                BITONICSORT512_512();

                // Set minValue to the new min in the warp register queue
                // Registers are sorted in descending order k0 > k1 > k2 ... > k_n
                minValue = k15;

                // Shift members in shared memory to beginning
                bufferSize         -= 256;
                if (tgx < bufferSize)
                {
                    psKey[tgx]      = psKey[tgx + 256];
                    psValue[tgx]    = psValue[tgx + 256];
                }
            }

            // Advance to next block of data
            rpos                    += cData._warpSize;
        }

        // Do final sort if buffer has any remaining data
        if ((bufferSize > 0) || (width <= 256))
        {
            // Store sentinel values in registers
            k8                       = -MAX_VALUE;
            k9                       = -MAX_VALUE;
            k10                      = -MAX_VALUE;
            k11                      = -MAX_VALUE;
            k12                      = -MAX_VALUE;
            k13                      = -MAX_VALUE;
            k14                      = -MAX_VALUE;
            k15                      = -MAX_VALUE;
            v8                       = 0;
            v9                       = 0;
            v10                      = 0;
            v11                      = 0;
            v12                      = 0;
            v13                      = 0;
            v14                      = 0;
            v15                      = 0;

            // Load last block of unsorted data into registers
            if (tgx < bufferSize)
            {
                k8                   = psKey[tgx];
                v8                   = psValue[tgx];
            }
            if (tgx + cData._warpSize < bufferSize)
            {
                k9                   = psKey[tgx + cData._warpSize];
                v9                   = psValue[tgx + cData._warpSize];
            }
            if (tgx + 2 * cData._warpSize < bufferSize)
            {
                k10                  = psKey[tgx + 2 * cData._warpSize];
                v10                  = psValue[tgx + 2 * cData._warpSize];
            }
            if (tgx + 3 * cData._warpSize < bufferSize)
            {
                k11                  = psKey[tgx + 3 * cData._warpSize];
                v11                  = psValue[tgx + 3 * cData._warpSize];
            }
            if (tgx + 4 * cData._warpSize < bufferSize)
            {
                k12                  = psKey[tgx + 4 * cData._warpSize];
                v12                  = psValue[tgx + 4 * cData._warpSize];
            }
            if (tgx + 5 * cData._warpSize < bufferSize)
            {
                k13                  = psKey[tgx + 5 * cData._warpSize];
                v13                  = psValue[tgx + 5 * cData._warpSize];
            }  
            if (tgx + 6 * cData._warpSize < bufferSize)
            {
                k14                  = psKey[tgx + 6 * cData._warpSize];
                v14                  = psValue[tgx + 6 * cData._warpSize];
            }
            if (tgx + 7 * cData._warpSize < bufferSize)
            {
                k15                  = psKey[tgx + 7 * cData._warpSize];
                v15                  = psValue[tgx + 7 * cData._warpSize];
            } 

            BITONICSORT512_512();
        }

        // Copy results to key and value pointers
        NNFloat* pKey                = pKeyBuffer + pos * k;
        uint32_t* pValue             = pValueBuffer + pos * k;                
        wpos                         = tgx;
        if (wpos < k)
        {
            pKey[wpos]               = k8;
            pValue[wpos]             = v8;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k9;
            pValue[wpos]             = v9;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k10;
            pValue[wpos]             = v10;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k11;
            pValue[wpos]             = v11;
        }
        if (wpos < k)
        {
            pKey[wpos]               = k12;
            pValue[wpos]             = v12;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k13;
            pValue[wpos]             = v13;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k14;
            pValue[wpos]             = v14;
        }
        wpos                        += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]               = k15;
            pValue[wpos]             = v15;
        }
    }
}



void kCalculateTopK(NNFloat* pOutput, NNFloat *pKey, uint32_t* pValue, uint32_t batch, uint32_t width, uint32_t k)
{
    uint32_t blocks                 = (batch + 3) / 4;
    if (k <= 32)
    {
        kCalculateTopK_32_kernel<<<blocks, 128>>>(pOutput, pKey, pValue, batch, width, k);
        LAUNCHERROR("kCalculateTopK_32_kernel");
    }
    else if (k <= 64)
    {
        kCalculateTopK_64_kernel<<<blocks, 128>>>(pOutput, pKey, pValue, batch, width, k);
        LAUNCHERROR("kCalculateTopK_64_kernel");
    }
    else if (k <= 128)
    {
        kCalculateTopK_128_kernel<<<blocks, 128>>>(pOutput, pKey, pValue, batch, width, k);
        LAUNCHERROR("kCalculateTopK_128_kernel");
    }
    else
    {
        kCalculateTopK_256_kernel<<<blocks, 128>>>(pOutput, pKey, pValue, batch, width, k);
        LAUNCHERROR("kCalculateTopK_256_kernel");
    }
}


__global__ void
LAUNCH_BOUNDS()
kCalculateTopK_kernel(NNFloat* pOutputKey, NNFloat* pOutputValue, NNFloat* pKeyBuffer, NNFloat* pValueBuffer, uint32_t batch, uint32_t width, uint32_t k)
{
__shared__ volatile NNFloat sKey[160 * 4];
__shared__ volatile NNFloat sValue[160 * 4];


    uint32_t pos                    = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                    = threadIdx.x & cData._warpMask;
        
    
    if (pos < batch)
    {
        pOutputKey                 += pos * width;
        pOutputValue               += pos * width;
        uint32_t offset             = threadIdx.x >> cData._warpBits;
        volatile NNFloat* psKey     = &sKey[160 * offset];
        volatile NNFloat* psValue   = &sValue[160 * offset];

        // Initialize values to 
        NNFloat k0                  = -MAX_VALUE;
        NNFloat k1                  = -MAX_VALUE;
        NNFloat k2                  = -MAX_VALUE;
        NNFloat k3                  = -MAX_VALUE;
        NNFloat k4                  = -MAX_VALUE;
        NNFloat k5                  = -MAX_VALUE;
        NNFloat k6                  = -MAX_VALUE;
        NNFloat k7                  = -MAX_VALUE;
        NNFloat v0                  = 0.0f;
        NNFloat v1                  = 0.0f;
        NNFloat v2                  = 0.0f;
        NNFloat v3                  = 0.0f;
        NNFloat v4                  = 0.0f;
        NNFloat v5                  = 0.0f;
        NNFloat v6                  = 0.0f;
        NNFloat v7                  = 0.0f;

        // Read first 128 elements into registers
        uint32_t wpos               = tgx;
        if (wpos < width)
        {
            k0                      = pOutputKey[wpos];
            v0                      = pOutputValue[wpos];
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k1                      = pOutputKey[wpos];
            v1                      = pOutputValue[wpos];
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k2                      = pOutputKey[wpos];
            v2                      = pOutputValue[wpos];
        }
        wpos                       += cData._warpSize;
        if (wpos < width)
        {
            k3                      = pOutputKey[wpos];
            v3                      = pOutputValue[wpos];
        }
  
        // Run through remainder of data
        NNFloat minValue            = -MAX_VALUE;
        uint32_t rpos               = 128;
        uint32_t bufferSize         = 0;
        NNFloat key1, key2;
        NNFloat value1, value2;
        uint32_t otgx;
        bool flag;
        while (rpos < width)
        {
            // Read block of data
            unsigned wpos           = rpos + tgx;
            NNFloat key             = -MAX_VALUE;
            NNFloat value           = 0.0f;
            if (wpos < width)
            {
                key                 = pOutputKey[wpos];
                value               = pOutputValue[wpos];              
            }
            
            // Add values > minValue to shared memory buffer
            uint32_t count          = BALLOT(key > minValue);
            if (key > minValue)
            {
                uint32_t mask       = 0xffffffff >> (32 - tgx);
                uint32_t offset     = __popc(count & mask);
                offset             += bufferSize;
                psKey[offset]       = key;
                psValue[offset]     = value;
            }
            bufferSize             += __popc(count);

            // Check if buffer is full
            if (bufferSize >= 128)
            {
                // Sort 256 elements
                k4                  = psKey[tgx];
                v4                  = psValue[tgx];
                k5                  = psKey[tgx + cData._warpSize];
                v5                  = psValue[tgx + cData._warpSize];
                k6                  = psKey[tgx + 2 * cData._warpSize];
                v6                  = psValue[tgx + 2 * cData._warpSize];
                k7                  = psKey[tgx + 3 * cData._warpSize];
                v7                  = psValue[tgx + 3 * cData._warpSize];
                bool flag;
                BITONICSORT256_256();

                // Shift members in shared memory to beginning
                bufferSize         -= 128;
                if (tgx < bufferSize)
                {
                    psKey[tgx]      = psKey[tgx + 128];
                    psValue[tgx]    = psValue[tgx + 128];
                }
            }

            // Advance to next block of data
            rpos                   += cData._warpSize;
        }

        // Do final sort if buffer has any remaining data
        if ((bufferSize > 0) || (width <= 128))
        {
            // Store sentinel values in registers
            k4                      = -MAX_VALUE;
            k5                      = -MAX_VALUE;
            k6                      = -MAX_VALUE;
            k7                      = -MAX_VALUE;
            v4                      = 0;
            v5                      = 0;
            v6                      = 0;
            v7                      = 0;
            
            // Load last block of unsorted data into registers
            if (tgx < bufferSize)
            {
                k4                  = psKey[tgx];
                v4                  = psValue[tgx];
            }
            if (tgx + cData._warpSize < bufferSize)
            {
                k5                  = psKey[tgx + cData._warpSize];
                v5                  = psValue[tgx + cData._warpSize];
            }
            if (tgx  + 2 * cData._warpSize < bufferSize)
            {
                k6                  = psKey[tgx + 2 * cData._warpSize];
                v6                  = psValue[tgx + 2 * cData._warpSize];
            }
            if (tgx + 3 * cData._warpSize < bufferSize)
            {          
                k7                  = psKey[tgx + 3 * cData._warpSize];
                v7                  = psValue[tgx + 3 * cData._warpSize];
            }
            BITONICSORT256_256();
        }

        // Copy results to index and value pointers
        NNFloat* pKey               = pKeyBuffer + pos * k;
        NNFloat* pValue             = pValueBuffer + pos * k;                
        wpos                        = tgx;
        if (wpos < k)
        {
            pKey[wpos]              = k0;
            pValue[wpos]            = v0;
        }
        wpos                       += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]              = k1;
            pValue[wpos]            = v1;
        }
        wpos                       += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]              = k2;
            pValue[wpos]            = v2;
        }
        wpos                       += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]              = k3;
            pValue[wpos]            = v3;
        }
    }
}

void kCalculateTopK(NNFloat* pOutputKey, NNFloat* pOutputValue, NNFloat *pKey, NNFloat* pValue, uint32_t batch, uint32_t width, uint32_t k)
{
    uint32_t blocks                 = (batch + 3) / 4;
    kCalculateTopK_kernel<<<blocks, 128>>>(pOutputKey, pOutputValue, pKey, pValue, batch, width, k);
    LAUNCHERROR("kCalculateTopK_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateTopK_kernel(NNFloat* pOutputKey, uint32_t* pOutputValue, NNFloat* pKeyBuffer, uint32_t* pValueBuffer, uint32_t batch, uint32_t width, uint32_t k)
{
__shared__ volatile NNFloat sKey[160 * 4];
__shared__ volatile uint32_t sValue[160 * 4];
    uint32_t pos                        = (blockIdx.x * blockDim.x + threadIdx.x) >> cData._warpBits;
    uint32_t tgx                        = threadIdx.x & cData._warpMask;        
    
    if (pos < batch)
    {
        pOutputKey                     += pos * width;
        pOutputValue                   += pos * width;
        uint32_t offset                 = threadIdx.x >> cData._warpBits;
        volatile NNFloat* psKey         = &sKey[160 * offset];
        volatile uint32_t* psValue      = &sValue[160 * offset];

        // Initialize values to 
        NNFloat k0                      = -MAX_VALUE;
        NNFloat k1                      = -MAX_VALUE;
        NNFloat k2                      = -MAX_VALUE;
        NNFloat k3                      = -MAX_VALUE;
        NNFloat k4                      = -MAX_VALUE;
        NNFloat k5                      = -MAX_VALUE;
        NNFloat k6                      = -MAX_VALUE;
        NNFloat k7                      = -MAX_VALUE;
        uint32_t v0                     = 0;
        uint32_t v1                     = 0;
        uint32_t v2                     = 0;
        uint32_t v3                     = 0;
        uint32_t v4                     = 0;
        uint32_t v5                     = 0;
        uint32_t v6                     = 0;
        uint32_t v7                     = 0;

        // Read first 128 elements into registers
        uint32_t wpos                   = tgx;
        if (wpos < width)
        {
            k0                              = pOutputKey[wpos];
            v0                              = pOutputValue[wpos];
        }
        wpos                               += cData._warpSize;
        if (wpos < width)
        {
            k1                              = pOutputKey[wpos];
            v1                              = pOutputValue[wpos];
        }
        wpos                               += cData._warpSize;
        if (wpos < width)
        {
            k2                              = pOutputKey[wpos];
            v2                              = pOutputValue[wpos];
        }
        wpos                               += cData._warpSize;
        if (wpos < width)
        {
            k3                              = pOutputKey[wpos];
            v3                              = pOutputValue[wpos];
        }
     
        // Run through remainder of data
        NNFloat minValue                    = -MAX_VALUE;
        uint32_t rpos                       = 128;
        uint32_t bufferSize                 = 0;
        NNFloat key1, key2;
        uint32_t value1, value2;
        uint32_t otgx;
        bool flag;
        while (rpos < width)
        {
            // Read block of data
            unsigned wpos                   = rpos + tgx;
            NNFloat key                     = -MAX_VALUE;
            NNFloat value                   = 0.0f;
            if (wpos < width)
            {
                key                         = pOutputKey[wpos];
                value                       = pOutputValue[wpos];              
            }
            
            // Add values > minValue to shared memory buffer
            uint32_t count                  = BALLOT(key > minValue);
            if (key > minValue)
            {
                uint32_t mask               = 0xffffffff >> (32 - tgx);
                uint32_t offset             = __popc(count & mask);
                offset                     += bufferSize;
                psKey[offset]               = key;
                psValue[offset]             = value;
            }
            bufferSize                     += __popc(count);

            // Check if buffer is full
            if (bufferSize >= 128)
            {
                // Sort 256 elements
                k4                          = psKey[tgx];
                v4                          = psValue[tgx];
                k5                          = psKey[tgx + cData._warpSize];
                v5                          = psValue[tgx + cData._warpSize];
                k6                          = psKey[tgx + 2 * cData._warpSize];
                v6                          = psValue[tgx + 2 * cData._warpSize];
                k7                          = psKey[tgx + 3 * cData._warpSize];
                v7                          = psValue[tgx + 3 * cData._warpSize];
                bool flag;
                BITONICSORT256_256();

                // Shift members in shared memory to beginning
                bufferSize                 -= 128;
                if (tgx < bufferSize)
                {
                    psKey[tgx]              = psKey[tgx + 128];
                    psValue[tgx]            = psValue[tgx + 128];
                }
            }

            // Advance to next block of data
            rpos                           += cData._warpSize;
        }

        // Do final sort if buffer has any remaining data
        if ((bufferSize > 0) || (width <= 128))
        {
            // Store sentinel values in registers
            k4                              = -MAX_VALUE;
            k5                              = -MAX_VALUE;
            k6                              = -MAX_VALUE;
            k7                              = -MAX_VALUE;
            v4                              = 0;
            v5                              = 0;
            v6                              = 0;
            v7                              = 0;

            // Load last block of unsorted data into registers
            if (tgx < bufferSize)
            {
                k4                          = psKey[tgx];
                v4                          = psValue[tgx];
            }
            if (tgx + cData._warpSize < bufferSize)
            {
                k5                          = psKey[tgx + cData._warpSize];
                v5                          = psValue[tgx + cData._warpSize];
            }
            if (tgx + 2 * cData._warpSize < bufferSize)
            {
                k6                          = psKey[tgx + 2 * cData._warpSize];
                v6                          = psValue[tgx + 2 * cData._warpSize];
            }
            if (tgx + 3 * cData._warpSize < bufferSize)
            {
                k7                          = psKey[tgx + 3 * cData._warpSize];
                v7                          = psValue[tgx + 3 * cData._warpSize];
            }

            BITONICSORT256_256();
        }

        // Copy results to index and value pointers
        NNFloat* pKey                       = pKeyBuffer + pos * k;
        uint32_t* pValue                    = pValueBuffer + pos * k;                
        wpos                                = tgx;
        if (wpos < k)
        {
            pKey[wpos]                      = k0;
            pValue[wpos]                    = v0;
        }
        wpos                               += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]                      = k1;
            pValue[wpos]                    = v1;
        }
        wpos                               += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]                      = k2;
            pValue[wpos]                    = v2;
        }
        wpos                               += cData._warpSize;
        if (wpos < k)
        {
            pKey[wpos]                      = k3;
            pValue[wpos]                    = v3;
        }
    }
}


void kCalculateTopK(NNFloat* pOutputKey, uint32_t* pOutputValue, NNFloat *pKey, uint32_t * pValue, uint32_t batch, uint32_t width, uint32_t k)
{
    uint32_t blocks                         = (batch + 3) / 4;
    kCalculateTopK_kernel<<<blocks, 128>>>(pOutputKey, pOutputValue, pKey, pValue, batch, width, k);
    LAUNCHERROR("kCalculateTopK_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kNormalizeWeights_kernel(NNFloat norm, uint32_t outputStride, uint32_t inputStride, NNFloat* pWeight)
{
    uint32_t pos                            = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < outputStride)
    {
        NNFloat r2                          = 0.0f;
        NNFloat* pEnd                       = pWeight + outputStride * inputStride; 
        pWeight                            += pos;
        NNFloat* p                          = pWeight;
        
        // Calculate squared weight vector length
        while (p < pEnd)
        {
            NNFloat x                       = *p;
            r2                             += x * x;
            p                              += outputStride;
        } 
        
        // Normalize if necessary
        if (r2 > norm * norm)
        {
            norm                           *= rsqrt(r2);
            p                               = pWeight;
            while (p < pEnd)
            {
                *p                         *= norm;
                p                          += outputStride;
            }             
        }
    }

}

void kNormalizeWeights(NNFloat norm, uint32_t outputStride, uint32_t inputStride, NNFloat* pWeight)
{
    uint32_t blocks                         = (outputStride + 127) / 128;
    kNormalizeWeights_kernel<<<blocks, 128>>>(norm, outputStride, inputStride, pWeight); 
    LAUNCHERROR("kNormalizeWeights_kernel");   
}


__global__ void
LAUNCH_BOUNDS()
kCalculateWeightMagnitudes_kernel(uint32_t outputStride, uint32_t inputStride, NNFloat* pWeight, NNFloat* pMagnitude)
{
    uint32_t pos                            = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < outputStride)
    {
        NNFloat r2                          = 0.0f;
        NNFloat* pEnd                       = pWeight + outputStride * inputStride; 
        pWeight                            += pos;
        NNFloat* p                          = pWeight;
        
        // Calculate squared weight vector length
        while (p < pEnd)
        {
            NNFloat x                       = *p;
            r2                             += x * x;
            p                              += outputStride;
        } 
        
        // Output to accumulator
        pMagnitude[pos]                     = r2;
    }

}

void kCalculateWeightMagnitudes(uint32_t outputStride, uint32_t inputStride, NNFloat* pWeight, NNFloat* pMagnitude)
{
    uint32_t blocks                         = (outputStride + 127) / 128;
    kCalculateWeightMagnitudes_kernel<<<blocks, 128>>>(outputStride, inputStride, pWeight, pMagnitude); 
    LAUNCHERROR("kCalculateWeightMagnitudes_kernel");   
}

__global__ void
LAUNCH_BOUNDS()
kNormalizeWeightMagnitudes_kernel(NNFloat norm, uint32_t outputStride, uint32_t inputStride, NNFloat* pWeight, NNFloat* pMagnitude)
{
    uint32_t pos                            = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < outputStride)
    {
        NNFloat r2                          = pMagnitude[pos];
        NNFloat* pEnd                       = pWeight + outputStride * inputStride; 
        pWeight                            += pos;
        NNFloat* p                          = pWeight;
        
        // Normalize if necessary
        if (r2 > norm * norm)
        {
            norm                           *= rsqrt(r2);
            p                               = pWeight;
            while (p < pEnd)
            {
                *p                         *= norm;
                p                          += outputStride;
            }             
        }
    }

}

void kNormalizeWeightMagnitudes(NNFloat norm, uint32_t outputStride, uint32_t inputStride, NNFloat* pWeight, NNFloat* pMagnitude)
{
    uint32_t blocks                         = (outputStride + 127) / 128;
    kNormalizeWeightMagnitudes_kernel<<<blocks, 128>>>(norm, outputStride, inputStride, pWeight, pMagnitude); 
    LAUNCHERROR("kNormalizeWeightMagnitudes_kernel");   
}

__global__ void
LAUNCH_BOUNDS()
kCalculateScaledBiasedDropout_kernel(NNFloat* pUnit, NNFloat* pRandom, NNFloat p, NNFloat target, NNFloat a, NNFloat b, size_t size)
{
    uint64_t pos                            = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat r                           = pRandom[pos];
        pUnit[pos]                          = (r < p) ? target : a * pUnit[pos] + b;
    }
}

void kCalculateScaledBiasedDropout(NNFloat* pUnit, NNFloat* pRandom, uint32_t batch, uint32_t stride, NNFloat p, NNFloat target, NNFloat a, NNFloat b)
{
    hiprandGenerateUniform(getGpu()._RNG, pRandom, batch * stride);
    unsigned long blocks                = CalculateBlocks(batch * stride);
    kCalculateScaledBiasedDropout_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pRandom, p, a * target + b, a, b, batch * stride);
    LAUNCHERROR("kCalculateScaledBiasedDropout_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kCalculateDropout_kernel(NNFloat* pUnit, NNFloat* pRandom, NNFloat p, NNFloat scale, NNFloat target, size_t size)
{
    uint64_t pos                            = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat r                           = pRandom[pos];
        pUnit[pos]                          = (r < p) ? target : scale * pUnit[pos];
    }
}

void kCalculateDropout(NNFloat* pUnit, NNFloat* pRandom, uint32_t batch, uint32_t stride, NNFloat p, NNFloat target)
{
    hiprandGenerateUniform(getGpu()._RNG, pRandom, batch * stride);
    unsigned long blocks                = CalculateBlocks(batch * stride);
    NNFloat scale                       = (target == (NNFloat)0.0) ? (NNFloat)1.0 / ((NNFloat)1.0 - p) : (NNFloat)1.0;
    kCalculateDropout_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pUnit, pRandom, p, scale, target, batch * stride);
    LAUNCHERROR("kCalculateDropout_kernel");
}

__global__ void 
LAUNCH_BOUNDS()
kCalculateMaxout_kernel(NNFloat* pSrc, size_t size, NNFloat* pDst)
{
    uint64_t pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
    {
        NNFloat s = pSrc[pos];
        NNFloat d = pDst[pos];
        if (s > d)
            pDst[pos]                   = s;
    }
}

void kCalculateMaxout(NNFloat* pSrc, size_t size, NNFloat* pDst)
{
    unsigned long blocks                = CalculateBlocks(size);
    kCalculateMaxout_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pSrc, size, pDst);
    LAUNCHERROR("kCalculateMaxout_kernel");
}

__global__ void 
LAUNCH_BOUNDS()
kCalculateCosine_kernel(NNFloat* pVector1, NNFloat* pVector2, uint32_t stride, NNFloat* pDPOut, NNFloat* pAOut, NNFloat* pBOut, uint32_t outStride)
{
__shared__ NNFloat sDP[64];     // Shared memory accumulator between warps
__shared__ NNFloat sA[64];      // Shared memory accumulator between warps
__shared__ NNFloat sB[64];      // Shared memory accumulator between warps


    // Preincrement pointers
    pVector1               += blockIdx.x * stride + threadIdx.x;
    pVector2               += blockIdx.x * stride + threadIdx.x;
    pDPOut                 += blockIdx.x * outStride;
    pAOut                  += blockIdx.x * outStride;
    pBOut                  += blockIdx.x * outStride;    
    uint32_t pos            = threadIdx.x;
    NNFloat dp              = (NNFloat)0;
    NNFloat al              = (NNFloat)0;
    NNFloat bl              = (NNFloat)0;
    
    // Calculate running sum
    while (pos < stride)
    {
        NNFloat a           = *pVector1;
        NNFloat b           = *pVector2;
        dp                 += a * b;
        al                 += a * a;
        bl                 += b * b;
        pVector1           += blockDim.x;
        pVector2           += blockDim.x;
        pos                += blockDim.x;
    }
    
    
    // Reduce results within warps
    uint32_t tgx            = threadIdx.x & cData._warpMask;
    dp                     += SHFL(dp, tgx ^ 1);
    al                     += SHFL(al, tgx ^ 1);
    bl                     += SHFL(bl, tgx ^ 1);
    dp                     += SHFL(dp, tgx ^ 2);
    al                     += SHFL(al, tgx ^ 2);
    bl                     += SHFL(bl, tgx ^ 2);
    dp                     += SHFL(dp, tgx ^ 4);
    al                     += SHFL(al, tgx ^ 4);
    bl                     += SHFL(bl, tgx ^ 4);
    dp                     += SHFL(dp, tgx ^ 8);
    al                     += SHFL(al, tgx ^ 8);
    bl                     += SHFL(bl, tgx ^ 8);    
    dp                     += SHFL(dp, tgx ^ 16); 
    al                     += SHFL(al, tgx ^ 16);
    bl                     += SHFL(bl, tgx ^ 16);
    if (tgx == 0)           
    {
        uint32_t index      = threadIdx.x >> cData._warpBits;
        sDP[index]          = dp;
        sA[index]           = al;
        sB[index]           = bl;
    }
    __syncthreads();
    
    // Reduce results between warps
    if (threadIdx.x < cData._warpSize)
    {
        uint32_t limit      = (blockDim.x + cData._warpSize -1) >> cData._warpBits;
        al                  = (threadIdx.x < limit) ? sA[threadIdx.x]     : (NNFloat)0;      
        bl                  = (threadIdx.x < limit) ? sB[threadIdx.x]     : (NNFloat)0; 
        dp                  = (threadIdx.x < limit) ? sDP[threadIdx.x]    : (NNFloat)0;
        dp                 += SHFL(dp, tgx ^ 1);
        al                 += SHFL(al, tgx ^ 1);
        bl                 += SHFL(bl, tgx ^ 1);
        dp                 += SHFL(dp, tgx ^ 2);
        al                 += SHFL(al, tgx ^ 2);
        bl                 += SHFL(bl, tgx ^ 2);
        dp                 += SHFL(dp, tgx ^ 4);
        al                 += SHFL(al, tgx ^ 4);
        bl                 += SHFL(bl, tgx ^ 4);
        dp                 += SHFL(dp, tgx ^ 8);
        al                 += SHFL(al, tgx ^ 8);
        bl                 += SHFL(bl, tgx ^ 8);    
        dp                 += SHFL(dp, tgx ^ 16); 
        al                 += SHFL(al, tgx ^ 16);
        bl                 += SHFL(bl, tgx ^ 16);        
                         
        
        // Write final sum
        if (threadIdx.x == 0)
        {
            al              = sqrt(al) + (NNFloat)1.0e-08;
            bl              = sqrt(bl) + (NNFloat)1.0e-08;
            dp             /= al * bl;
            *pAOut          = al;
            *pBOut          = bl;
            *pDPOut         = dp;
            //printf("COS %d %f %f %f\n", blockIdx.x, dp, al, bl);
        }
    }
} 

// Calculates cosine and saves vector lengths for future gradient calculation
void kCalculateCosine(NNFloat* pVector1In, NNFloat* pVector2In, uint32_t batch, uint32_t stride, NNFloat* pDPOut, NNFloat* pAOut, NNFloat* pBOut, uint32_t outStride)
{
    unsigned long threads = max(32, min(stride, getGpu()._threadsPerBlock));
    kCalculateCosine_kernel<<<batch, threads>>>(pVector1In, pVector2In, stride, pDPOut, pAOut, pBOut, outStride);
    LAUNCHERROR("kCalculateCosine_kernel");    
}



__global__ void 
LAUNCH_BOUNDS()
kCalculateDotProduct_kernel(NNFloat* pVector1In, NNFloat* pVector2In, uint32_t strideIn, NNFloat* pDPOut, uint32_t strideOut)
{
__shared__ NNFloat sDP[32];     // Shared memory accumulator between warps

    // Preincrement pointers
    pVector1In             += blockIdx.x * strideIn + threadIdx.x;
    pVector2In             += blockIdx.x * strideIn + threadIdx.x;
    pDPOut                 += blockIdx.x * strideOut;
    uint32_t pos            = threadIdx.x;
    NNFloat dp              = (NNFloat)0;

    
    // Calculate running sum
    while (pos < strideIn)
    {
        NNFloat a           = *pVector1In;
        NNFloat b           = *pVector2In;
        dp                 += a * b;
        pVector1In         += blockDim.x;
        pVector2In         += blockDim.x;
        pos                += blockDim.x;
    }
    
    
    // Reduce results within warps
    REDUCE(dp)
    uint32_t tgx            = threadIdx.x & cData._warpMask;    
    if (tgx == 0)           
    {
        uint32_t index      = threadIdx.x >> cData._warpBits;
        sDP[index]          = dp;
    }
    __syncthreads();
    
    // Reduce results between warps
    if (threadIdx.x < cData._warpSize)
    {
        uint32_t limit      = (blockDim.x + cData._warpSize -1) >> cData._warpBits;
        dp                  = (threadIdx.x < limit) ? sDP[threadIdx.x]    : (NNFloat)0;
        REDUCE(dp)                 
        
        // Write final sum
        if (threadIdx.x == 0)
        {
            *pDPOut         = dp;      
        }
    }
} 

// Calculates dot product
void kCalculateDotProduct(NNFloat* pVector1In, NNFloat* pVector2In, uint32_t batch, uint32_t strideIn, NNFloat* pDPOut, uint32_t strideOut)
{
    unsigned long threads = max(32, min(strideIn, getGpu()._threadsPerBlock));
    kCalculateDotProduct_kernel<<<batch, threads>>>(pVector1In, pVector2In, strideIn, pDPOut, strideOut);
    LAUNCHERROR("kCalculateDotProduct_kernel");    
}

#include "cub/util_allocator.cuh"
#include "cub/device/device_radix_sort.cuh"

template<typename KeyType, typename ValueType> size_t kInitSort(uint32_t items, GpuBuffer<KeyType>* pbKey, GpuBuffer<ValueType>* pbValue)
{
    uint32_t itemStride                     = ((items + 511) >> 9) << 9;
    size_t tempBytes;
    hipcub::DoubleBuffer<KeyType> d_keys(pbKey->_pDevData, pbKey->_pDevData + itemStride);
    hipcub::DoubleBuffer<ValueType> d_values(pbValue->_pDevData, pbValue->_pDevData + itemStride);
    hipcub::DeviceRadixSort::SortPairs(NULL, tempBytes, d_keys, d_values, items);
    return tempBytes;
}

template<typename KeyType, typename ValueType> bool kSort(uint32_t items, KeyType* pKey0, KeyType* pKey1, ValueType* pValue0, ValueType* pValue1, char* pTemp, size_t tempBytes)
{
    hipcub::DoubleBuffer<KeyType>  d_keys(pKey0, pKey1);
    hipcub::DoubleBuffer<ValueType> d_values(pValue0, pValue1);
    hipcub::DeviceRadixSort::SortPairs(pTemp, tempBytes, d_keys, d_values, items);
    return true;   
}

__global__ void
LAUNCH_BOUNDS()
kAddScaleBuffers_kernel(NNFloat* pDst, NNFloat* pSrc, NNFloat scale, uint64_t size)
{
    uint64_t pos                            = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
        *(pDst + pos)                      += *(pSrc + pos) * scale;
}

void kAddScaleBuffers(NNFloat* pDst, NNFloat* pSrc, NNFloat scale, uint64_t size)
{
    uint32_t blocks                         = CalculateBlocks(size);
    kAddScaleBuffers_kernel<<<blocks, getGpu()._threadsPerBlock>>>(pDst, pSrc, scale, size);
    LAUNCHERROR("kAddScaleBuffers_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAddBuffers_kernel(NNFloat* pDst, NNFloat* pSrc, uint64_t size)
{
    uint64_t pos                            = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < size)
        *(pDst + pos)                      += *(pSrc + pos);
}

void kAddBuffers(NNFloat* pDst, NNFloat* pSrc, uint64_t size, hipStream_t stream)
{
    // Skip zero-sized copies
    if (size == 0)
        return;

    uint32_t blocks                         = CalculateBlocks(size);
    kAddBuffers_kernel<<<blocks, getGpu()._threadsPerBlock, 0, stream>>>(pDst, pSrc, size);
    LAUNCHERROR("kAddBuffers_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kAddBuffers2D_kernel(NNFloat* pDst, uint32_t dpitch, NNFloat* pSrc, uint32_t spitch, uint32_t width)
{
    uint64_t yOffset                        = blockIdx.y * blockDim.x + threadIdx.x;
    if (yOffset < width)
    {
        uint64_t dpos                       = blockIdx.x * dpitch + yOffset;
        uint64_t spos                       = blockIdx.x * spitch + yOffset;
        pDst[dpos]                         += pSrc[spos];
    }
}

void kAddBuffers2D(NNFloat* pDst, uint32_t dpitch, NNFloat* pSrc, uint32_t spitch, uint32_t width, uint32_t height, hipStream_t stream)
{
    // Bail if nothing to add
    if ((height == 0) || (width == 0))
        return;

    dim3 grid(height, (width + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);       
    kAddBuffers2D_kernel<<<grid, getGpu()._threadsPerBlock, 0, stream>>>(pDst, dpitch, pSrc, spitch, width);
    LAUNCHERROR("kAddBuffers2D_kernel");
}

__global__ void
LAUNCH_BOUNDS()
kCopy2D_kernel(NNFloat* pDst, uint32_t dpitch, NNFloat* pSrc, uint32_t spitch, uint32_t width)
{
    uint64_t yOffset                        = blockIdx.y * blockDim.x + threadIdx.x;
    if (yOffset < width)
    {
        uint64_t dpos                       = blockIdx.x * dpitch + yOffset;
        uint64_t spos                       = blockIdx.x * spitch + yOffset;
        pDst[dpos]                          = pSrc[spos];
    }
}

void kCopy2D(NNFloat* pDst, uint32_t dpitch, NNFloat* pSrc, uint32_t spitch, uint32_t width, uint32_t height, hipStream_t stream)
{
    // Bail if nothing to copy
    if ((height == 0) || (width == 0))
        return;
    dim3 grid(height, (width + getGpu()._threadsPerBlock - 1) / getGpu()._threadsPerBlock);      
    kCopy2D_kernel<<<grid, getGpu()._threadsPerBlock, 0, stream>>>(pDst, dpitch, pSrc, spitch, width);
    LAUNCHERROR("kCopy2D_kernel");
}

// Instantiates allowable templated functions so we can hide the implementations here
// instead of in the header file because we're mixing CUDA and C++ and that's
// a migraine headache in the making otherwise.
template size_t kInitSort<NNFloat, NNFloat>  (uint32_t, GpuBuffer<NNFloat>*, GpuBuffer<NNFloat>*);
template size_t kInitSort<uint32_t, NNFloat> (uint32_t, GpuBuffer<uint32_t>*, GpuBuffer<NNFloat>*);
template size_t kInitSort<NNFloat, uint32_t> (uint32_t, GpuBuffer<NNFloat>*, GpuBuffer<uint32_t>*);
template size_t kInitSort<uint32_t, uint32_t>(uint32_t, GpuBuffer<uint32_t>*, GpuBuffer<uint32_t>*);

template bool kSort<NNFloat, NNFloat>(uint32_t, NNFloat*, NNFloat*, NNFloat*, NNFloat*, char*, size_t);
template bool kSort<NNFloat, uint32_t>(uint32_t, NNFloat*, NNFloat*, uint32_t*, uint32_t*, char*, size_t);
template bool kSort<uint32_t, NNFloat>(uint32_t, uint32_t*, uint32_t*, NNFloat*, NNFloat*, char*, size_t);
template bool kSort<uint32_t, uint32_t>(uint32_t, uint32_t*, uint32_t*, uint32_t*, uint32_t*, char*, size_t);

#define EXPLICITLY_INSTANTIATE_KERNELS(T)                                                                                                                                                       \
template void kLoadSparseAnalogDenoisedInputUnit<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*);                                           \
template void kLoadIndexedSparseAnalogDenoisedInputUnit<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*);                         \
template void kLoadSparseAnalogInputUnit<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*);                                                             \
template void kLoadIndexedSparseAnalogInputUnit<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*);                                           \
template void kCalculateSparseAnalogZ<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*, NNFloat);                                             \
template void kCalculateIndexedSparseAnalogZ<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*, NNFloat);                           \
template void kCalculateSparseAnalogDenoisedZ<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*, NNFloat*, NNFloat);                           \
template void kCalculateIndexedSparseAnalogDenoisedZ<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*, NNFloat*, NNFloat);         \
template void kCalculateSparseTransposedAnalogMatrix<T>(uint32_t, uint32_t, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, uint32_t*, uint32_t*, NNFloat*);                                     \
template void kCalculateIndexedSparseTransposedAnalogMatrix<T>(uint32_t, uint32_t, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, uint32_t*, uint32_t*, NNFloat*);                   \
template void kCalculateSparseTransposedAnalogDenoisedMatrix<T>(uint32_t, uint32_t, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*, uint32_t*, uint32_t*, NNFloat*);                   \
template void kCalculateIndexedSparseTransposedAnalogDenoisedMatrix<T>(uint32_t, uint32_t, uint32_t*, uint64_t*, uint64_t*, uint32_t*, NNFloat*, T*, NNFloat*, uint32_t*, uint32_t*, NNFloat*); \
template void kLoadInputUnit<T>(uint32_t, uint32_t, uint32_t, NNFloat*, T*);                                                                                                                    \
template void kLoadIndexedInputUnit<T>(uint32_t, uint32_t, uint32_t, NNFloat*, uint32_t*, T*);




                                                                                 \
/**/

EXPLICITLY_INSTANTIATE_KERNELS(NNFloat)
EXPLICITLY_INSTANTIATE_KERNELS(double)
EXPLICITLY_INSTANTIATE_KERNELS(unsigned char)
EXPLICITLY_INSTANTIATE_KERNELS(char)
EXPLICITLY_INSTANTIATE_KERNELS(uint32_t)
EXPLICITLY_INSTANTIATE_KERNELS(uint64_t)
EXPLICITLY_INSTANTIATE_KERNELS(int32_t)
EXPLICITLY_INSTANTIATE_KERNELS(int64_t)
